#include "hip/hip_runtime.h"
// filter.cu

#include <algorithm>
#include <chrono>
#include <exception>
#include <future>
#include <iostream>
#include <limits>
#include <numeric>
#include <optional>
#include <span>
#include <thread>
#include <tuple>
#include <utility> // pair
#include <hip/hip_runtime.h>
#include "filter.cuh"
#include "stream-data.h"
#include "merge-filter-data.h"
#include "util.h"

//#define LOGGING

namespace cm {

namespace {

/*
__device__ bool variation_indices_shown = false;

__device__ void print_variation_indices(
    const device::VariationIndices* __restrict__ variation_indices) {
  if (variation_indices_shown)
    return;
  printf("device:\n");
  for (int s{}; s < kNumSentences; ++s) {
    const auto& vi = variation_indices[s];
    for (index_t v{}; v < vi.num_variations; ++v) {
      const auto n = vi.num_combo_indices[v];
      printf("sentence %d, variation %d, indices: %d\n", s, v, n);
    }
  }
  variation_indices_shown = true;
}
*/

template <typename T> __device__ __forceinline__ T load(const T* addr) {
  return *(const volatile T*)addr;
}

template <typename T> __device__ __forceinline__ void store(T* addr, T val) {
  *(volatile T*)addr = val;
}

// Test if the supplied source contains both of the primary sources described
// by any of the supplied source descriptor pairs.
__device__ bool source_contains_any_descriptor_pair(
    const SourceCompatibilityData& source,
    const UsedSources::SourceDescriptorPair* __restrict__ src_desc_pairs,
    const unsigned num_src_desc_pairs) {

  __shared__ bool contains_both;
  if (!threadIdx.x) contains_both = false;
  // one thread per src_desc_pair
  for (unsigned idx{}; idx * blockDim.x < num_src_desc_pairs; ++idx) {
    __syncthreads();
    if (contains_both) return true;
    const auto pair_idx = idx * blockDim.x + threadIdx.x;
    if (pair_idx < num_src_desc_pairs) {
      if (source.usedSources.has(src_desc_pairs[pair_idx])) {
        contains_both = true;
      }
    }
  }
  return false;
}

__global__ void get_compatible_sources_kernel(
    const SourceCompatibilityData* __restrict__ sources,
    const unsigned num_sources,
    const UsedSources::
        SourceDescriptorPair* __restrict__ incompatible_src_desc_pairs,
    const unsigned num_src_desc_pairs,
    compat_src_result_t* __restrict__ results) {
  // one block per source
  for (unsigned idx{blockIdx.x}; idx < num_sources; idx += gridDim.x) {
    const auto& source = sources[idx];
    // I don't understand why this is required, but fails synccheck and
    // produces wobbly results without.
    __syncthreads();
    if (!source_contains_any_descriptor_pair(
          source, incompatible_src_desc_pairs, num_src_desc_pairs)) {
      if (!threadIdx.x) {
        results[idx] = 1;
      }
    }
  }
}

extern __shared__ unsigned dynamic_shared[];

/*
__device__ bool is_source_OR_compatible_old(const SourceCompatibilityData& source,
    const unsigned num_or_args,
    const device::OrSourceData* __restrict__ or_arg_sources,
    const unsigned num_or_arg_sources) {
  //  extern __shared__ result_t or_arg_results[];
  // ASSUMPTION: # of --or args will always be smaller than block size.
  if (threadIdx.x < num_or_args) {
    or_arg_results[threadIdx.x] = 0;
  }
  const auto chunk_size = blockDim.x;
  const auto chunk_max = num_or_arg_sources;
  for (unsigned chunk_idx{}; chunk_idx * chunk_size < chunk_max; ++chunk_idx) {
    __syncthreads();
    const auto or_arg_src_idx = chunk_idx * chunk_size + threadIdx.x;
    if (or_arg_src_idx < num_or_arg_sources) {
      const auto& or_src = or_arg_sources[or_arg_src_idx];
      // NB! order of a,b in a.isOrCompat(b) here matters!
      if (or_src.src.isOrCompatibleWith(source)) {
        or_arg_results[or_src.or_arg_idx] = 1;
      }
    }
  }
  __syncthreads();
  if (!threadIdx.x) {
    bool compat_with_all{true};
    for (int i{}; i < num_or_args; ++i) {
      if (!or_arg_results[i]) {
        compat_with_all = false;
        break;
      }
    }
    return compat_with_all;
  }
  return false;
}

  if (num_or_args > 0) {
    // source must also be OR compatible with at least one source
    // of each or_arg
    if (is_source_OR_compatible(
            source, num_or_args, or_arg_sources, num_or_arg_sources)) {
      is_or_compat = true;
    }
    __syncthreads();
    if (!is_or_compat) {
      continue;
    }
  }
*/

struct SmallestSpansResult {
  bool skip;
  ComboIndexSpanPair spans;
};
using smallest_spans_result_t = SmallestSpansResult;

// variation_indices is an optimization. it allows us to restrict comparisons
// of a candidate source to only those xor_sources that have the same (or no)
// variation for each sentence - since a variation mismatch will alway result
// in comparison failure.
//
// individual xor_src indices are potentially (and often) duplicated in the
// variation_indices lists. for example, if a particular compound xor_src has
// variations S1:V1 and S2:V3, its xor_src_idx will appear in the variation
// indices lists for both of sentences 1 and 2.
//
// because of this, we only need to compare a candidate source with the
// xor_sources that have the same (or no) variations for a *single* sentence.
//
// the question then becomes, which sentence should we choose?
//
// the answer: the one with the fewest indices! (which results in the fewest
// comparisons). that's what this function determines.
//
__device__ smallest_spans_result_t get_smallest_src_index_spans(
    const SourceCompatibilityData& source,
    const device::VariationIndices* __restrict__ variation_indices) {
  index_t fewest_indices{std::numeric_limits<index_t>::max()};
  int sentence_with_fewest{-1};
  for (int s{}; s < kNumSentences; ++s) {
    // if there are no xor_sources that contain a primary source from this
    // sentence, skip it. (it would result in num_indices == all_indices which
    // is the worst case).
    const auto& vi = variation_indices[s];
    if (!vi.num_variations) continue;

    // if the candidate source has no primary source from this sentence, skip
    // it. (same reason as above).
    const auto variation = source.usedSources.variations[s] + 1;
    if (!variation) continue;

    // sum the xor_src indices that have no variation (index 0), with those
    // that have the same variation as the candidate source, for this sentence.
    // remember the sentence with the smallest sum.
    const auto num_indices =
        vi.num_combo_indices[0] + vi.num_combo_indices[variation];
    if (num_indices < fewest_indices) {
      fewest_indices = num_indices;
      sentence_with_fewest = s;
      if (!fewest_indices) break;
    }
  }
  if (sentence_with_fewest < 0) {
    // there are no sentences from which both the candidate source and any
    // xor_source contain a primary source. we can skip xor-compat checks
    // since they will all succeed.
    return {true};
  }
  if (!fewest_indices) {
    // both the candidate source and all xor_sources contain a primary source
    // from sentence_with_fewest, but all xor_sources use a different variation
    // than the candidate source. we can skip all xor-compat checks since they
    // will all fail due to variation mismatch.
    return {true};
  }
  const auto variation =
      source.usedSources.variations[sentence_with_fewest] + 1;
  const auto& vi = variation_indices[sentence_with_fewest];
  return {false,  //
      std::make_pair(vi.get_index_span(0), vi.get_index_span(variation))};
}

__device__ uint64_t get_flat_idx(
    uint64_t block_idx, unsigned thread_idx = threadIdx.x) {
  return block_idx * uint64_t(blockDim.x) + thread_idx;
}

__device__ auto get_xor_combo_index(
    uint64_t flat_idx, const ComboIndexSpanPair& idx_spans) {
  if (flat_idx < idx_spans.first.size()) return idx_spans.first[flat_idx];
  flat_idx -= idx_spans.first.size();
  assert(flat_idx < idx_spans.second.size());
  return idx_spans.second[flat_idx];
}

__device__ auto get_combo_index(uint64_t flat_idx,
    const index_t* __restrict__ idx_list_sizes, unsigned num_idx_lists) {
  combo_index_t combo_idx{};
  for (int list_idx{int(num_idx_lists) - 1}; list_idx >= 0; --list_idx) {
    const auto idx_list_size = idx_list_sizes[list_idx];
    if (combo_idx) combo_idx *= idx_list_size;
    combo_idx += flat_idx % idx_list_size;
    flat_idx /= idx_list_size;
  }
  return combo_idx;
}

const int kXorChunkIdx = 0;
const int kOrChunkIdx = 1;
const int kXorResultsIdx = 2;
const int kNumSharedIndices = 4;  // 32-bit align

// TODO: templatize this & is_source_XOR_compatible
// Test if a source is OR compatible with the OR source specified by the
// supplied combo index and index lists.
__device__ bool is_source_OR_compatible(const SourceCompatibilityData& source,
    combo_index_t or_combo_idx,
    const SourceCompatibilityData* __restrict__ or_src_lists,
    const index_t* __restrict__ or_src_list_start_indices,
    const index_t* __restrict__ or_idx_lists,
    const index_t* __restrict__ or_idx_list_start_indices,
    const index_t* __restrict__ or_idx_list_sizes, unsigned num_or_idx_lists) {
  for (unsigned list_idx{}; list_idx < num_or_idx_lists; ++list_idx) {
    const auto or_src_list =
        &or_src_lists[or_src_list_start_indices[list_idx]];
    const auto idx_list = &or_idx_lists[or_idx_list_start_indices[list_idx]];
    const auto idx_list_size = or_idx_list_sizes[list_idx];
    const auto or_src_idx = idx_list[or_combo_idx % idx_list_size];
    const auto& or_src = or_src_list[or_src_idx];
    if (!or_src.isOrCompatibleWith(source)) return false;
    or_combo_idx /= idx_list_size;
  }
  return true;
}

// Get the next block-sized chunk of OR sources and test them for variation-
// compatibililty with the XOR source specified by the supplied xor_combo_index,
// and for OR-compatibility with the supplied source.
// Return true if at least one OR source is compatible.
__device__ bool get_next_OR_sources_chunk(const SourceCompatibilityData& source,
    unsigned or_chunk_idx, const combo_index_t xor_combo_idx,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes, unsigned num_idx_lists,
    const SourceCompatibilityData* __restrict__ or_src_lists,
    const index_t* __restrict__ or_src_list_start_indices,
    const index_t* __restrict__ or_idx_lists,
    const index_t* __restrict__ or_idx_list_start_indices,
    const index_t* __restrict__ or_idx_list_sizes,
    const unsigned num_or_idx_lists, const unsigned num_or_sources,
    result_t* __restrict__ or_results) {
  /*
  //  __shared__ bool is_variation_compat;
  //  __shared__ bool is_or_compat;
  if (!threadIdx.x) {
    is_or_compat = false;
    //    is_variation_compat = false;
    or_results[threadIdx.x] = 0;
  }
  __syncthreads();
  */
  const auto or_flat_idx = get_flat_idx(or_chunk_idx);
  if (or_flat_idx < num_or_sources) { // or: if (>=) return false;
    const auto or_combo_idx =
        get_combo_index(or_flat_idx, or_idx_list_sizes, num_or_idx_lists);
#if 0
    if (are_variations_compatible(
            or_combo_idx, or_src_lists, xor_combo_idx, xor_src_lists)) {
#endif
    // is_variation_compat = true;
    // }
    // }

    // TODO: get rid of sync here. and is_variation_compat shared var.
    // if a *specific instance* of an OR-source combo_idx is variation-
    // compatible with XOR-source, we can immediately check it for
    // compatibility with a *specfiic instance* of an OR source combo-idx.
    // In other words, nested if statements here, vs. sync, and only one
    // is_or_compat shared. *I THINK*.

    //__syncthreads();
    // if (is_variation_compat) {
    if (is_source_OR_compatible(source, or_combo_idx, or_src_lists,
            or_src_list_start_indices, or_idx_lists, or_idx_list_start_indices,
            or_idx_list_sizes, num_or_idx_lists)) {
      //is_or_compat = true;
      or_results[threadIdx.x] = 1;
      return true;
    }
  }
  //__syncthreads();
  //  return is_or_compat;
  return false;
}

// for one XOR result
__device__ bool get_next_compatible_OR_sources(
    const SourceCompatibilityData& source, unsigned or_chunk_idx,
    combo_index_t xor_combo_idx,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes, unsigned num_xor_idx_lists,
    const SourceCompatibilityData* __restrict__ or_src_lists,
    const index_t* __restrict__ or_src_list_start_indices,
    const index_t* __restrict__ or_idx_lists,
    const index_t* __restrict__ or_idx_list_start_indices,
    const index_t* __restrict__ or_idx_list_sizes,
    const unsigned num_or_idx_lists, const unsigned num_or_sources,
    const combo_index_t* __restrict__ or_variations,
    const unsigned num_or_variations, result_t* __restrict__ or_results) {
  const auto block_size = blockDim.x;
  unsigned* or_chunk_idx_ptr = &dynamic_shared[kOrChunkIdx];
  __shared__ bool any_or_compat;
  if (!threadIdx.x) any_or_compat = false;
  __syncthreads();
  for (; or_chunk_idx * block_size < num_or_sources; ++or_chunk_idx) {
    if (get_next_OR_sources_chunk(source, or_chunk_idx, xor_combo_idx,
            xor_src_lists, xor_src_list_start_indices, xor_idx_lists,
            xor_idx_list_start_indices, xor_idx_list_sizes, num_xor_idx_lists,
            or_src_lists, or_src_list_start_indices, or_idx_lists,
            or_idx_list_start_indices, or_idx_list_sizes, num_or_idx_lists,
            num_or_sources, or_results)) {
      any_or_compat = true;
    }
    __syncthreads();
    if (any_or_compat) break;
  }
  if (!threadIdx.x) *or_chunk_idx_ptr = or_chunk_idx + 1;
  return any_or_compat;
}

__device__ auto next_xor_result_idx(
    unsigned result_idx, const result_t* __restrict__ xor_results) {
  const auto block_size = blockDim.x;
  while ((result_idx < block_size) && !xor_results[result_idx])
    result_idx++;
  return result_idx;
}

// for all XOR results
__device__ bool get_next_compatible_OR_sources(
    const SourceCompatibilityData& source, unsigned or_chunk_idx,
    unsigned xor_chunk_idx, const result_t* __restrict__ xor_results,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes, unsigned num_xor_idx_lists,
    const ComboIndexSpanPair& idx_spans,
    const SourceCompatibilityData* __restrict__ or_src_lists,
    const index_t* __restrict__ or_src_list_start_indices,
    const index_t* __restrict__ or_idx_lists,
    const index_t* __restrict__ or_idx_list_start_indices,
    const index_t* __restrict__ or_idx_list_sizes,
    const unsigned num_or_idx_lists, const unsigned num_or_sources,
    const combo_index_t* __restrict__ or_variations,
    const unsigned num_or_variations, result_t* __restrict__ or_results) {
  const auto block_size = blockDim.x;
  unsigned* xor_results_idx_ptr = &dynamic_shared[kXorResultsIdx];
  unsigned* or_chunk_idx_ptr = &dynamic_shared[kOrChunkIdx];
  __shared__ bool any_or_compat;
  if (!threadIdx.x) {
    if (!or_chunk_idx) {
      *xor_results_idx_ptr = next_xor_result_idx(0, xor_results);
    }
    any_or_compat = false;
  }
  __syncthreads();
  auto xor_results_idx = *xor_results_idx_ptr;
  //
  // TODO: I think I could use a local here for results_idx and
  // update it at end of loop.
  //
  while (xor_results_idx < block_size) {
    const auto xor_flat_idx = get_flat_idx(xor_chunk_idx, xor_results_idx);
    const auto xor_combo_idx = get_xor_combo_index(xor_flat_idx, idx_spans);
    if (get_next_compatible_OR_sources(source, or_chunk_idx, xor_combo_idx,
            xor_src_lists, xor_src_list_start_indices, xor_idx_lists,
            xor_idx_list_start_indices, xor_idx_list_sizes, num_xor_idx_lists,
            or_src_lists, or_src_list_start_indices, or_idx_lists,
            or_idx_list_start_indices, or_idx_list_sizes, num_or_idx_lists,
            num_or_sources, or_variations, num_or_variations, or_results)) {
      any_or_compat = true;
    }
    __syncthreads();
    // sync failure here? using local for results_idx eliminates problem
    // (and a __syncthreads)
    if (any_or_compat) break; 
    //if (!threadIdx.x) {
      xor_results_idx = next_xor_result_idx(xor_results_idx + 1, xor_results);
      //}
      //__syncthreads();
  }
  if (!threadIdx.x) {
    // If we incremented xor_results_idx beyond the end of the xor_results
    // array at the end of the loop above, manually zero the or_chunk_idx.
    //if (*xor_results_idx >= block_size) *or_chunk_idx_ptr = 0;
    // TODO:
    *xor_results_idx_ptr = (xor_results_idx < block_size) ? xor_results_idx : 0;
  }
  return any_or_compat;
}

// Test if a source is XOR compatible with the XOR source specified by the
// supplied combo index and index lists.
__device__ bool is_source_XOR_compatible(const SourceCompatibilityData& source,
    combo_index_t combo_idx,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes,
    unsigned num_xor_idx_lists) {

  for (unsigned list_idx{}; list_idx < num_xor_idx_lists; ++list_idx) {
    const auto xor_src_list =
        &xor_src_lists[xor_src_list_start_indices[list_idx]];
    const auto idx_list = &xor_idx_lists[xor_idx_list_start_indices[list_idx]];
    const auto idx_list_size = xor_idx_list_sizes[list_idx];
    const auto xor_src_idx = idx_list[combo_idx % idx_list_size];
    if (!source.isXorCompatibleWith(xor_src_list[xor_src_idx])) return false;
    combo_idx /= idx_list_size;
  }
  return true;
}

// Get the next block-sized chunk of XOR sources and test them for 
// compatibility with the supplied source.
// Return true if at least one XOR source is compatible.
__device__ bool get_next_XOR_sources_chunk(
    const SourceCompatibilityData& source, const unsigned xor_chunk_idx,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes,
    const unsigned num_xor_idx_lists, const ComboIndexSpanPair& idx_spans,
    result_t* __restrict__ xor_results) {
  const auto num_xor_indices = idx_spans.first.size() + idx_spans.second.size();
  __shared__ bool is_xor_compat;
  if (!threadIdx.x) {
    is_xor_compat = false;
    xor_results[threadIdx.x] = 0;
  }
  __syncthreads();
  //
  // TODO: i have a sneaking suspicious i could get rid of the shared/syncs
  // in here similar to get_next_OR_sources_chunk
  //
  const auto xor_flat_idx = get_flat_idx(xor_chunk_idx);
  if (xor_flat_idx < num_xor_indices) {
    const auto xor_combo_idx = get_xor_combo_index(xor_flat_idx, idx_spans);
    if (is_source_XOR_compatible(source, xor_combo_idx, xor_src_lists,
            xor_src_list_start_indices, xor_idx_lists,
            xor_idx_list_start_indices, xor_idx_list_sizes,
            num_xor_idx_lists)) {
      is_xor_compat = true;
      xor_results[threadIdx.x] = 1;
    }
  }
  __syncthreads();
  return is_xor_compat;
}

// Loop through block-sized chunks of XOR sources until we find one that 
// contains at leasst one XOR source that is compatibile with the supplied
// source, or until all XOR sources are exhausted.
// Return true if at least one XOR source is compatible.
__device__ bool get_next_compatible_XOR_sources(
    const SourceCompatibilityData& source, unsigned xor_chunk_idx,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes,
    const unsigned num_xor_idx_lists, const ComboIndexSpanPair& idx_spans,
    result_t* __restrict__ xor_results) {
  const auto block_size = blockDim.x;
  const auto num_indices = idx_spans.first.size() + idx_spans.second.size();
  unsigned* xor_chunk_idx_ptr = &dynamic_shared[kXorChunkIdx];
  __shared__ bool any_xor_compat;
  if (!threadIdx.x) any_xor_compat = false;
  __syncthreads();
  for (; xor_chunk_idx * block_size < num_indices; ++xor_chunk_idx) {
    if (get_next_XOR_sources_chunk(source, xor_chunk_idx, xor_src_lists,
            xor_src_list_start_indices, xor_idx_lists,
            xor_idx_list_start_indices, xor_idx_list_sizes, num_xor_idx_lists,
            idx_spans, xor_results)) {
      any_xor_compat = true;
    }
    __syncthreads();
    if (any_xor_compat) break;
  }
  if (!threadIdx.x) *xor_chunk_idx_ptr = xor_chunk_idx + 1;
  return any_xor_compat;
}

// Test if the supplied source is:
// * XOR-compatible with any of the supplied XOR sources
// * OR-compatible with any of the supplied OR sources which are
//   variation-compatible with the compatible XOR source.
//
// In other words:
// For each XOR source that is XOR-compatible with Source
//   For each OR source that is variation-compatible with XOR source
//     If OR source is OR-compatible with Source
//       is_compat = true;
__device__ bool is_compat_loop(const SourceCompatibilityData& source,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes,
    const unsigned num_xor_idx_lists, const ComboIndexSpanPair& idx_spans,
    const SourceCompatibilityData* __restrict__ or_src_lists,
    const index_t* __restrict__ or_src_list_start_indices,
    const index_t* __restrict__ or_idx_lists,
    const index_t* __restrict__ or_idx_list_start_indices,
    const index_t* __restrict__ or_idx_list_sizes,
    const unsigned num_or_idx_lists, const unsigned num_or_sources,
    const combo_index_t* __restrict__ or_variations,
    const unsigned num_or_variations) {
  const auto block_size = blockDim.x;
  unsigned* xor_chunk_idx_ptr = &dynamic_shared[kXorChunkIdx];
  unsigned* or_chunk_idx_ptr = &dynamic_shared[kOrChunkIdx];
  result_t* xor_results = (result_t*)&dynamic_shared[kNumSharedIndices];
  result_t* or_results = (result_t*)&xor_results[block_size];
  __shared__ bool any_xor_compat;
  __shared__ bool any_or_compat;
  const unsigned num_xor_indices =
      idx_spans.first.size() + idx_spans.second.size();
  if (!threadIdx.x) {
    *xor_chunk_idx_ptr = 0;
    *or_chunk_idx_ptr = 0;
    // TODO: might do better with an enum here, fewer back-to-back __syncthreads
    // calls potentially
    any_xor_compat = false;
    any_or_compat = false;
  }
  __syncthreads();
  for (;;) {
    if (!any_xor_compat) {
      if (*xor_chunk_idx_ptr * block_size >= num_xor_indices) break;
      if (get_next_compatible_XOR_sources(source, *xor_chunk_idx_ptr,
              xor_src_lists, xor_src_list_start_indices, xor_idx_lists,
              xor_idx_list_start_indices, xor_idx_list_sizes, num_xor_idx_lists,
              idx_spans, xor_results)) {
        any_xor_compat = true;
      }
    }
    __syncthreads();
    if (any_xor_compat && !num_or_sources) return true;
#if 1
    if (any_xor_compat && !any_or_compat) {
      if (*or_chunk_idx_ptr * block_size >= num_or_sources) break;
      if (get_next_compatible_OR_sources(source, *or_chunk_idx_ptr,
              *xor_chunk_idx_ptr, xor_results, xor_src_lists,
              xor_src_list_start_indices, xor_idx_lists,
              xor_idx_list_start_indices, xor_idx_list_sizes, num_xor_idx_lists,
              idx_spans, or_src_lists, or_src_list_start_indices, or_idx_lists,
              or_idx_list_start_indices, or_idx_list_sizes, num_or_idx_lists,
              num_or_sources, or_variations, num_or_variations, or_results)) {
        any_or_compat = true;
      }
    }
    __syncthreads();
    if (any_or_compat) return true;
#endif
  }
  return false;
}

// explain better:
// Find sources that are:
// * XOR compatible with any of the supplied XOR sources, and
// * OR compatible with any of the supplied OR sources, which must in turn be
// * variation-compatible with the XOR source.
//
// Find compatible XOR source -> compare with variation-compatible OR sources.
__global__ void filter_kernel(
    const SourceCompatibilityData* __restrict__ src_list,
    const unsigned num_sources,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes,
    const unsigned num_xor_idx_lists,
    const device::VariationIndices* __restrict__ xor_variation_indices,
    const SourceIndex* __restrict__ src_indices,
    const index_t* __restrict__ src_list_start_indices,
    const compat_src_result_t* __restrict__ compat_src_results,
    const SourceCompatibilityData* __restrict__ or_src_lists,
    const index_t* __restrict__ or_src_list_start_indices,
    const index_t* __restrict__ or_idx_lists,
    const index_t* __restrict__ or_idx_list_start_indices,
    const index_t* __restrict__ or_idx_list_sizes,
    const unsigned num_or_idx_lists, const unsigned num_or_sources,
    const combo_index_t* __restrict__ or_variations,
    const unsigned num_or_variations, result_t* __restrict__ results,
    int stream_idx) {
  const auto block_size = blockDim.x;
  // TODO __global__ compat int, cuda_memset to zero before launching kernel
  //result_t* compat_results = (result_t*)&dynamic_shared[kNumSharedIndices];
  __shared__ bool is_compat;
  if (!threadIdx.x) is_compat = false;
  // for each source (one block per source)
  for (unsigned idx{blockIdx.x}; idx < num_sources; idx += gridDim.x) {
    __syncthreads();
    const auto src_idx = src_indices[idx];
    const auto flat_idx =
        src_list_start_indices[src_idx.listIndex] + src_idx.index;
    if (compat_src_results && !compat_src_results[flat_idx]) continue;
    const auto& source = src_list[flat_idx];
    auto smallest = get_smallest_src_index_spans(source, xor_variation_indices);
    if (smallest.skip) continue;
    if (is_compat_loop(source, xor_src_lists, xor_src_list_start_indices,
            xor_idx_lists, xor_idx_list_start_indices, xor_idx_list_sizes,
            num_xor_idx_lists, smallest.spans, or_src_lists,
            or_src_list_start_indices, or_idx_lists, or_idx_list_start_indices,
            or_idx_list_sizes, num_or_idx_lists, num_or_sources, or_variations,
            num_or_variations)) {
      is_compat = true;
    }
    __syncthreads();
    if (is_compat && !threadIdx.x) {
      results[src_idx.listIndex] = 1;
      is_compat = false;
    }
  }
}

}  // anonymous namespace

void run_filter_kernel(int threads_per_block, StreamData& stream,
    const MergeFilterData& mfd, const SourceCompatibilityData* device_src_list,
    const compat_src_result_t* device_compat_src_results,
    result_t* device_results, const index_t* device_list_start_indices) {
  int num_sm;
  hipDeviceGetAttribute(&num_sm, hipDeviceAttributeMultiprocessorCount, 0);
  int threads_per_sm;;
  hipDeviceGetAttribute(&threads_per_sm, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);

  stream.is_running = true;
  stream.sequence_num = StreamData::next_sequence_num();

  auto block_size = threads_per_block ? threads_per_block : 768;
  auto blocks_per_sm = threads_per_sm / block_size;
  assert(blocks_per_sm * block_size == threads_per_sm);
  auto grid_size = num_sm * blocks_per_sm;  // aka blocks per grid
  // xor_chunk_idx, or_chunk_idx, xor_result_idx, xor_results, or_results
  // results could probably be moved to global
  auto shared_bytes = 
      kNumSharedIndices * sizeof(unsigned) + block_size * 2 * sizeof(result_t);
  // ensure any async alloc/copies are complete on main thread stream
  hipError_t err = hipStreamSynchronize(hipStreamPerThread);
  assert_cuda_success(err, "run_filter_kernel sync");
  dim3 grid_dim(grid_size);
  dim3 block_dim(block_size);
  auto num_or_sources = util::sum_sizes(mfd.host_or.compat_idx_lists);
  //std::cerr << "num or sources: " << num_or_sources << std::endl;
  stream.xor_kernel_start.record();
  filter_kernel<<<grid_dim, block_dim, shared_bytes, stream.hip_stream>>>(
      // Sources
      device_src_list, stream.src_indices.size(),
      // XOR sources
      mfd.device_xor.src_lists, mfd.device_xor.src_list_start_indices,
      mfd.device_xor.idx_lists, mfd.device_xor.idx_list_start_indices,
      mfd.device_xor.idx_list_sizes, mfd.host_xor.compat_idx_lists.size(),
      mfd.device_xor.variation_indices,
      // Sources again
      stream.device_src_indices, device_list_start_indices,
      // XOR sources again i think, for count > 2
      device_compat_src_results,
      // OR sources
      mfd.device_or.src_lists, mfd.device_or.src_list_start_indices,
      mfd.device_or.idx_lists, mfd.device_or.idx_list_start_indices,
      mfd.device_or.idx_list_sizes, mfd.host_or.compat_idx_lists.size(),
      num_or_sources, mfd.device_or.combo_indices,
      mfd.host_or.combo_indices.size(), device_results, stream.stream_idx);
  stream.xor_kernel_stop.record();
  if constexpr (0) {
    std::cerr << "stream " << stream.stream_idx << " XOR kernel started with " << grid_size
              << " blocks of " << block_size << " threads" << std::endl;
  }
}

void run_get_compatible_sources_kernel(
    const SourceCompatibilityData* device_src_list, unsigned num_sources,
    const UsedSources::SourceDescriptorPair* device_src_desc_pairs,
    unsigned num_src_desc_pairs, compat_src_result_t* device_results) {
  int num_sm;
  hipDeviceGetAttribute(&num_sm, hipDeviceAttributeMultiprocessorCount, 0);
  int threads_per_sm;
  hipDeviceGetAttribute(
      &threads_per_sm, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
  auto block_size = 768;  // aka threads per block
  auto blocks_per_sm = threads_per_sm / block_size;
  assert(blocks_per_sm * block_size == threads_per_sm);
  auto grid_size = num_sm * blocks_per_sm;  // aka blocks per grid
  auto shared_bytes = 0;

  dim3 grid_dim(grid_size);
  dim3 block_dim(block_size);
  // async copies are on thread stream therefore auto synchronized
  hipStream_t stream = hipStreamPerThread;
  get_compatible_sources_kernel<<<grid_dim, block_dim, shared_bytes, stream>>>(
      device_src_list, num_sources, device_src_desc_pairs, num_src_desc_pairs,
      device_results);
}

}  // namespace cm
