#include "hip/hip_runtime.h"
// filter.cu

#include <algorithm>
#include <chrono>
#include <exception>
#include <future>
#include <limits>
#include <numeric>
#include <optional>
#include <thread>
#include <tuple>
#include <hip/hip_runtime.h>
#include "candidates.h"

//#define STREAM_LOG

namespace {

using namespace cm;

__device__ auto isSourceORCompatibleWithAnyOrSource(
  const SourceCompatibilityData& source, const OrSourceList& orSourceList) {
  auto compatible = false;
  for (const auto& orSource : orSourceList) {
    // skip any sources that were already determined to be XOR incompatible
    // or AND compatible with --xor sources.
    // Wait, what? why not "&& !andCompatible" ?
    if (!orSource.xorCompatible || orSource.andCompatible)
      continue;
    compatible = source.isOrCompatibleWith(orSource.source);
    if (compatible)
      break;
  }
  return compatible;
};

__device__ auto isSourceCompatibleWithEveryOrArg(
  const SourceCompatibilityData& source,
  const OrArgDataList& orArgDataList) {
  auto compatible = true;  // if no --or sources specified, compatible == true
  for (const auto& orArgData : orArgDataList) {
    // TODO: skip calls to here if container.compatible = true  which may have
    // been determined in Precompute phase @ markAllANDCompatibleOrSources()
    // and skip the XOR check as well in this case.
    compatible =
      isSourceORCompatibleWithAnyOrSource(source, orArgData.orSourceList);
    if (!compatible)
      break;
  }
  return compatible;
}

__device__ __host__ auto isSourceXORCompatibleWithAnyXorSource(
  const SourceCompatibilityData& source, const XorSource* xorSources,
  size_t numXorSources, int* compat_index = nullptr, int* reason = nullptr) {
  bool compatible = true;
  for (size_t i{}; i < numXorSources; ++i) {
    compatible = source.isXorCompatibleWith(xorSources[i], false, reason);
    if (compatible) {
      if (compat_index)
        *compat_index = i;
      break;
    }
  }
  return compatible;
}

using result_t = uint8_t;
using index_t = uint32_t;

struct SourceIndex {
  index_t listIndex{};
  index_t index{};

  bool operator<(const SourceIndex& rhs) const {
    return (listIndex < rhs.listIndex) || (index < rhs.index);
  }

  constexpr const char* as_string(char* buf) const {
    sprintf(buf, "%d:%d", listIndex, index);
    return buf;
  }
};

template <typename T> __device__ T load(const T* addr) {
  return *(const volatile T*)addr;
}

template <typename T> __device__ void store(T* addr, T val) {
  *(volatile T*)addr = val;
}

// one block per source
__global__ void xor_kernel_per_block(
  const SourceCompatibilityData* __restrict__ sources,
  const unsigned num_sources,
  const SourceCompatibilityData* __restrict__ xor_sources,
  const unsigned num_xor_sources,
  const SourceIndex* __restrict__ source_indices,
  const index_t* __restrict__ list_start_indices, result_t* results,
  int stream_idx) {
  //
  const auto threads_per_grid = gridDim.x * blockDim.x;
  // for each source (one block per source)
  for (unsigned idx{blockIdx.x}; idx < num_sources; idx += gridDim.x) {
    const auto src_idx = source_indices[idx];
    const auto flat_index =
      list_start_indices[src_idx.listIndex] + src_idx.index;
    const auto& source = sources[flat_index];
    auto& result = results[src_idx.listIndex];

    // for each xor_source (one block per xor source)
    for (unsigned xor_idx{threadIdx.x}; xor_idx < num_xor_sources;
         xor_idx += blockDim.x) {
      if (source.isXorCompatibleWith(xor_sources[xor_idx])) {
        store(&result, (uint8_t)1);
      }
      __syncthreads();
      if (load(&result))
        break;
    }
  }
}

// entire grid per source
__global__ void xor_kernel(const SourceCompatibilityData* __restrict__ sources,
  const unsigned num_sources,
  const SourceCompatibilityData* __restrict__ xor_sources,
  const unsigned num_xor_sources,
  const SourceIndex* __restrict__ source_indices,
  const index_t* __restrict__ list_start_indices, result_t* results,
  int stream_idx) {
  //
  const auto threads_per_grid = gridDim.x * blockDim.x;
  // for each source
  for (unsigned idx{}; idx < num_sources; ++idx) {
    const auto src_idx = source_indices[idx];
    const auto flat_index =
      list_start_indices[src_idx.listIndex] + src_idx.index;
    const auto& source = sources[flat_index];
    auto& result = results[src_idx.listIndex];

    // for each xor_source
    for (int start_idx = blockIdx.x * blockDim.x;
         start_idx + threadIdx.x < num_xor_sources;
         start_idx += threads_per_grid) {
      if (load(&result))
        break;
      if (!source.isXorCompatibleWith(xor_sources[start_idx + threadIdx.x]))
        continue;
      store(&result, (uint8_t)1);
    }
  }
}

struct IndexStates {
  enum class State
  {
    ready,
    compatible,
    done
  };

  struct Data {
    constexpr auto ready_state() const {
      return state == State::ready;
    }

    SourceIndex sourceIndex;
    State state = State::ready;
  };

  IndexStates() = delete;
  IndexStates(const SourceCompatibilityLists& sources) {
    list.resize(sources.size());  // i.e. "num_sourcelists"
    std::for_each(list.begin(), list.end(),
      [idx = 0](Data& data) mutable { data.sourceIndex.listIndex = idx++; });
    for (index_t list_start_index{}; const auto& sourceList : sources) {
      list_sizes.push_back(sourceList.size());
      list_start_indices.push_back(list_start_index);
      list_start_index += (index_t)sourceList.size();
    }
  }

  index_t flat_index(SourceIndex src_index) const {
    return list_start_indices.at(src_index.listIndex) + src_index.index;
  }

  auto list_size(index_t list_index) const {
    return list_sizes.at(list_index);
  }

  auto num_in_state(int first, int count, State state) const {
    int total{};
    for (int i{}; i < count; ++i) {
      if (list.at(first + i).state == state) {
        ++total;
      }
    }
    return total;
  }

  auto num_ready(int first, int count) const {
    return num_in_state(first, count, State::ready);
  }

  auto num_done(int first, int count) const {
    return num_in_state(first, count, State::done);
  }

  auto num_compatible(int first, int count) const {
    return num_in_state(first, count, State::compatible);
  }

  auto update(const std::vector<SourceIndex>& src_indices,
    const std::vector<result_t>& results,
    [[maybe_unused]] int stream_idx)  // for logging
  {
    constexpr static const bool logging = false;
    int num_compatible{};
    int num_done{};
    for (size_t i{}; i < src_indices.size(); ++i) {
      const auto src_idx = src_indices.at(i);
      auto& idx_state = list.at(src_idx.listIndex);
      const auto result = results.at(src_idx.listIndex);
      if (!idx_state.ready_state()) {
        continue;
      }
      if (result > 0) {
        idx_state.state = State::compatible;
        ++num_compatible;
      } else if (src_idx.index == list_sizes.at(src_idx.listIndex) - 1) {
        // if this is the result for the last source in a sourcelist,
        // mark the list (indexState) as done.
        idx_state.state = State::done;
        ++num_done;
      }
    }
#if 0
      std::cerr << "stream " << stream_idx
                << " update, total: " << src_indices.size()
                << ", compat: " << num_compatible
                << ", done: " << num_done << std::endl;
#endif
      return num_compatible;
  }

  auto get(index_t list_index) const {
    return list.at(list_index);
  }

  auto get_and_increment_index(index_t list_index)
    -> std::optional<SourceIndex> {
    auto& data = list.at(list_index);
    if (data.ready_state()
        && (data.sourceIndex.index < list_sizes.at(list_index))) {
      // capture and return value before increment
      auto capture = std::make_optional(data.sourceIndex);
      ++data.sourceIndex.index;
      return capture;
    }
    return std::nullopt;
  }

  int num_lists() const {
    return list.size();
  }

  auto get_next_fill_idx() {
    auto fill_idx = next_fill_idx;
    if (++next_fill_idx >= num_lists())
      next_fill_idx = 0;
    return fill_idx;
  }

  bool done{false};
  int next_fill_idx{0};
  std::vector<Data> list;
  std::vector<uint32_t> list_start_indices;
  std::vector<uint32_t> list_sizes;
};  // struct IndexStates

  //////////

std::vector<hipStream_t> streams;

// the pointers in this are allocated in device memory
struct KernelData {
private:
  using hr_time_point_t = decltype(std::chrono::high_resolution_clock::now());
  static const auto num_cores = 1280;
  static const auto max_chunks = 20ul;

public:
  static void init(
    std::vector<KernelData>& kernelVec, size_t num_sourcelists, size_t stride) {
    stride = std::min(num_sourcelists, stride);
    for (size_t i{}; i < kernelVec.size(); ++i) {
      auto& kernel = kernelVec.at(i);
      kernel.num_src_lists = num_sourcelists;
      kernel.num_list_indices = stride;
      kernel.source_indices.resize(kernel.num_list_indices);
      if (i >= streams.size()) {
        hipStream_t stream;
        hipError_t err = hipStreamCreate(&stream);
        assert((err == hipSuccess) && "failed to create stream");
        streams.push_back(stream);
      }
      kernel.stream_idx = i;
      kernel.stream = streams[i];
    }
  }

  static int next_sequence_num() {
    static int sequence_num{};
    return sequence_num++;
  }

  //

  int num_ready(const IndexStates& indexStates) const {
    return indexStates.num_ready(0, num_list_indices);
  }

  int num_done(const IndexStates& indexStates) const {
    return indexStates.num_done(0, num_list_indices);
  }

  int num_compatible(const IndexStates& indexStates) const {
    return indexStates.num_compatible(0, num_list_indices);
  }

  auto fillSourceIndices(IndexStates& idx_states, int max_idx) {
    source_indices.resize(idx_states.done ? 0 : max_idx);
    for (int idx{}; !idx_states.done && (idx < max_idx);) {
      auto num_skipped_idx{0};  // how many idx were skipped in a row
      // this loop logic is funky and brittle, but intentional
      for (auto list_idx = idx_states.get_next_fill_idx(); /*nada*/;
           list_idx = idx_states.get_next_fill_idx()) {
        const auto opt_src_idx = idx_states.get_and_increment_index(list_idx);
        if (opt_src_idx.has_value()) {
          const auto src_idx = opt_src_idx.value();
          assert(src_idx.listIndex == list_idx);
          source_indices.at(idx++) = src_idx;
          if (idx >= max_idx)
            break;
          num_skipped_idx = 0;
        } else if (++num_skipped_idx >= idx_states.num_lists()) {
          // we've skipped over the entire list (with index overlap)
          // and haven't consumed any indices. nothing left to do.
          idx_states.done = true;
          source_indices.resize(idx);
          break;
        }
      }
    }
#if 0
    std::cerr << "ending next_fill_idx: " << idx_states.next_fill_idx << std::endl;
    std::cerr << "stream " << stream_idx
              << " filled " << source_indices.size()
              << " of " << max_idx
              << ", first = " << (source_indices.empty() ? -1 : (int)source_indices.front().listIndex)
              << ", last = " << (source_indices.empty() ? -1 : (int)source_indices.back().listIndex)
              << ", done: " << std::boolalpha << idx_states.done
              << std::endl;
#endif
    return !source_indices.empty();
  }

  bool fillSourceIndices(IndexStates& idx_states) {
    return fillSourceIndices(idx_states, num_list_indices);
  }

  void allocCopy([[maybe_unused]] const IndexStates& idx_states) {
    hipError_t err = hipSuccess;
    auto indices_bytes = source_indices.size() * sizeof(SourceIndex);
    // alloc source indices
    if (!device_source_indices) {
      err =
        hipMallocAsync((void**)&device_source_indices, indices_bytes, stream);
      assert((err == hipSuccess) && "allocate source indices");
    }

    /*
    std::vector<index_t> flat_indices;
    flat_indices.reserve(source_indices.size());
    for (const auto& src_idx: source_indices) {
      flat_indices.push_back(idx_states.flat_index(src_idx));
    }
    */

    // copy source indices
    err = hipMemcpyAsync(device_source_indices, source_indices.data(),
      indices_bytes, hipMemcpyHostToDevice, stream);
    assert((err == hipSuccess) && "copy source indices");
  }

  auto hasWorkRemaining() const {
    return !source_indices.empty();
  }

  void dump() const {
    std::cerr << "kernel " << stream_idx << ", is_running: " << std::boolalpha
              << is_running << ", src_idx: " << source_indices.size()
              << std::endl;
  }

  int num_src_lists;  // total # of sourcelists (== # of device_results)
                      // (doesn't belong here)
  int num_list_indices;
  int stream_idx{-1};
  int sequence_num{};
  bool is_running{false};  // is running (may be complete; output not retrieved)
  bool has_run{false};     // has run at least once
  SourceIndex* device_source_indices{nullptr};  // in
  hipStream_t stream{nullptr};
  std::vector<SourceIndex> source_indices;  // .size() == num_results
  hr_time_point_t start_time;
};  // struct KernelData

//////////

struct ValueIndex {
  int value{};
  int index{-1};
};

auto anyWithWorkRemaining(const std::vector<KernelData>& kernelVec)
  -> std::optional<int> {
  for (size_t i{}; i < kernelVec.size(); ++i) {
    const auto& kernel = kernelVec[i];
    if (kernel.hasWorkRemaining()) {
      return std::make_optional(i);
    }
  }
  return std::nullopt;
}

bool anyIdleWithWorkRemaining(
  const std::vector<KernelData>& kernelVec, int& index) {
  for (size_t i{}; i < kernelVec.size(); ++i) {
    const auto& kernel = kernelVec[i];
    if (!kernel.is_running && kernel.hasWorkRemaining()) {
      index = i;
      return true;
    }
  }
  return false;
}

// TODO: std::optional, and above here
bool anyRunningComplete(const std::vector<KernelData>& kernelVec, int& index) {
  ValueIndex lowest = {std::numeric_limits<int>::max()};
  for (size_t i{}; i < kernelVec.size(); ++i) {
    const auto& kernel = kernelVec[i];
    if (kernel.is_running && (hipSuccess == hipStreamQuery(kernel.stream))) {
      if (kernel.sequence_num < lowest.value) {
        lowest.value = kernel.sequence_num;
        lowest.index = i;
      }
    }
  }
  if (lowest.index > -1) {
    index = lowest.index;
    return true;
  }
  return false;
}

bool get_next_available(std::vector<KernelData>& kernelVec, int& current) {
  using namespace std::chrono_literals;

  // First: ensure all primary streams have started at least once
  if (++current >= (int)kernelVec.size()) {
    current = 0;
  } else {
    const auto& kernel = kernelVec[current];
    if (!kernel.is_running && !kernel.has_run && kernel.hasWorkRemaining()) {
      return true;
    }
  }

  // Second: process results for any "running" stream that has completed
  if (anyRunningComplete(kernelVec, current)) {
    return true;
  }

  // Third: run any idle (non-running) stream with work remaining
  if (anyIdleWithWorkRemaining(kernelVec, current)) {
    return true;
  }

  // There is no idle stream, and no attachable running stream that has work
  // remaining. Is there any stream with work remaining? If not, we're done.
  if (!anyWithWorkRemaining(kernelVec).has_value()) {
    return false;
  }

  // Wait for one to complete.
  while (!anyRunningComplete(kernelVec, current)) {
    std::this_thread::sleep_for(5ms);
  }
  return true;
}

void run_xor_kernel(KernelData& kernel, int threads_per_block,
  const SourceCompatibilityData* device_sources, result_t* device_results,
  const index_t* device_list_start_indices) {
  //
  auto num_sm = 10;
  auto threads_per_sm = 2048;
  auto block_size = threads_per_block ? threads_per_block : 1024;
  auto blocks_per_sm = threads_per_sm / block_size;
  //  assert(blocks_per_sm * block_size == threads_per_sm);
  auto grid_size = num_sm * blocks_per_sm;  // aka blocks per grid
  auto shared_bytes = 0;

  kernel.is_running = true;
  kernel.sequence_num = KernelData::next_sequence_num();
  kernel.start_time = std::chrono::high_resolution_clock::now();
  dim3 grid_dim(grid_size);
  dim3 block_dim(block_size);
  xor_kernel_per_block<<<grid_dim, block_dim, shared_bytes, kernel.stream>>>(
    device_sources, kernel.source_indices.size(), PCD.device_xorSources,
    PCD.xorSourceList.size(), kernel.device_source_indices,
    device_list_start_indices, device_results, kernel.stream_idx);

#if 0 || defined(STREAM_LOG)
  std::cerr << "stream " << kernel.stream_idx
            << " started with " << grid_size << " blocks"
            << " of " << block_size << " threads"
          //<< " starting, sequence: " << kernel.sequence_num
            << std::endl;
#endif
}

// todo: kernel.getResults()
auto getKernelResults(KernelData& kernel, result_t* device_results) {
  hipError_t err = hipStreamSynchronize(kernel.stream);
  if (err != hipSuccess) {
    std::cerr << "Failed to synchronize, error: " << hipGetErrorString(err)
              << std::endl;
    assert((err == hipSuccess) && "sychronize");
  }

  // TODO this could go into kernelData
  std::vector<result_t> results(kernel.num_src_lists);
  auto results_bytes = kernel.num_src_lists * sizeof(result_t);
  err = hipMemcpyAsync(results.data(), device_results, results_bytes,
    hipMemcpyDeviceToHost, kernel.stream);
  if (err != hipSuccess) {
    std::cerr << "copy device results, error: " << hipGetErrorString(err)
              << std::endl;
    assert(!"copy results from device");
  }
  err = hipStreamSynchronize(kernel.stream);
  assert((err == hipSuccess) && "hipStreamSynchronize");
  kernel.is_running = false;
  return results;
}

auto count(const SourceCompatibilityLists& sources) {
  size_t num{};
  for (const auto& sourceList : sources) {
    num += sourceList.size();
  }
  return num;
}

auto* allocCopySources(const SourceCompatibilityLists& sources) {
  // alloc sources
  const hipStream_t stream = hipStreamPerThread;
  hipError_t err = hipSuccess;
  auto sources_bytes = count(sources) * sizeof(SourceCompatibilityData);
  SourceCompatibilityData* device_sources;
  err = hipMallocAsync((void**)&device_sources, sources_bytes, stream);
  assert((err == hipSuccess) && "failed to allocate sources");

  // copy sources
  size_t index{};
  for (const auto& sourceList : sources) {
    err = hipMemcpyAsync(&device_sources[index], sourceList.data(),
      sourceList.size() * sizeof(SourceCompatibilityData),
      hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) {
      fprintf(stdout, "failed to copy sources, error: %s", hipGetErrorString(err));
      throw std::runtime_error("copy sources");
    }
    index += sourceList.size();
  }
  return device_sources;
}

auto allocAndZeroResults(size_t num_results) {  // TODO hipStream_t stream) {
  hipStream_t stream = hipStreamPerThread;
  hipError_t err = hipSuccess;
  // alloc results
  auto results_bytes = num_results * sizeof(result_t);
  result_t* device_results;
  err = hipMallocAsync((void**)&device_results, results_bytes, stream);
  assert((err == hipSuccess) && "alloc results");
  err = hipMemsetAsync(device_results, results_bytes, 0, stream);
  assert((err == hipSuccess) && "zero results");
  return device_results;
}

auto* allocCopyListStartIndices(const IndexStates& index_states) {
  hipStream_t stream = hipStreamPerThread;
  hipError_t err = hipSuccess;
  // alloc indices
  auto indices_bytes = index_states.list_start_indices.size() * sizeof(index_t);
  index_t* device_indices;
  err = hipMallocAsync((void**)&device_indices, indices_bytes, stream);
  assert((err == hipSuccess) && "alloc list start indices");
  // copy indices
  err = hipMemcpyAsync(device_indices, index_states.list_start_indices.data(),
    indices_bytes, hipMemcpyHostToDevice, stream);
  assert((err == hipSuccess) && "copy list start indices");
  return device_indices;
}

auto flat_index(
  const SourceCompatibilityLists& sources, const SourceIndex src_idx) {
  uint32_t flat{};
  for (size_t i{}; i < src_idx.listIndex; ++i) {
    flat += sources.at(i).size();
  }
  return flat + src_idx.index;
}

void check(
  const SourceCompatibilityLists& sources, index_t list_index, index_t index) {
  constexpr const auto logging = true;
  if constexpr (logging) {
    SourceIndex src_idx{list_index, index};
    char idx_buf[32];
    char buf[64];
    snprintf(buf, sizeof(buf), "%s, flat: %d", src_idx.as_string(idx_buf),
      flat_index(sources, src_idx));
    auto& source = sources.at(list_index).at(index);
    source.dump(buf);
    int compat_index{-1};
    auto compat = isSourceXORCompatibleWithAnyXorSource(source,
      PCD.xorSourceList.data(), PCD.xorSourceList.size(), &compat_index);
    std::cerr << "compat: " << compat << " (" << compat_index << ")"
              << std::endl;
  }
}

void dump_xor(int index) {
  const XorSourceList& xorSources = PCD.xorSourceList;
  auto host_index = index;
  const auto& src = xorSources.at(host_index);
  char buf[32];
  snprintf(buf, sizeof(buf), "xor: device(%d) host(%d)", index, host_index);
  src.dump(buf);
}

int filter_task(int sum, int threads_per_block, int num_streams, int stride) {
  using namespace std::chrono;
  [[maybe_unused]] hipError_t err = hipSuccess;
  err = hipDeviceSetLimit(hipLimitPrintfFifoSize, 7'500'000);

  const auto& sources =
    allSumsCandidateData.find(sum)->second.sourceCompatLists;
  auto device_sources = allocCopySources(sources);
  IndexStates idx_states{sources};
  auto device_list_start_indices = allocCopyListStartIndices(idx_states);

  if (!num_streams)
    num_streams = 1;
  std::vector<KernelData> kernels(num_streams);
  stride = stride ? stride : 5000;
  KernelData::init(kernels, sources.size(), stride);

#if 0
  std::cerr << "sourcelists: " << sources.size() << ", streams: " << num_streams
            << ", stride: " << stride << std::endl;
#endif

  auto device_results = allocAndZeroResults(sources.size());

  int total_compat{};
  int current_kernel{-1};
  int actual_num_compat{};
  auto t0 = high_resolution_clock::now();
  while (get_next_available(kernels, current_kernel)) {
    auto& kernel = kernels.at(current_kernel);
    if (!kernel.is_running) {
      if (!kernel.fillSourceIndices(idx_states)) {
        kernel.is_running = false;
        continue;
      }
#if 0
      std::cerr << "stream " << kernel.stream_idx
                << " source_indices: " << kernel.source_indices.size()
                << ", ready: " << kernel.num_ready(idx_states)
                << std::endl;
#endif
      kernel.allocCopy(idx_states);
      run_xor_kernel(kernel, threads_per_block, device_sources, device_results,
        device_list_start_indices);
      continue;
    }

    kernel.has_run = true;
    kernel.is_running = false;

    auto results = getKernelResults(kernel, device_results);
    auto k1 = high_resolution_clock::now();
    auto d_kernel = duration_cast<milliseconds>(k1 - kernel.start_time).count();

    auto num_compat =
      idx_states.update(kernel.source_indices, results, kernel.stream_idx);
    total_compat += num_compat;

#if 0
    actual_num_compat = std::accumulate(results.begin(), results.end(), 0,
      [](int sum, result_t r) { return r ? sum + 1 : sum; });
    std::cerr << "stream " << kernel.stream_idx
              << " compat results: " << num_compat
              << " actual: " << actual_num_compat
              << " - results: " << d_results << "ms"
              << ", kernel: " << d_kernel << "ms"
              << std::endl;
#endif
  }
  auto t1 = high_resolution_clock::now();
  auto d_total = duration_cast<milliseconds>(t1 - t0).count();
  std::cerr << "sum(" << sum << ")"
            << " total compat: " << total_compat << " of "
            << sources.size() << " - " << d_total << "ms" << std::endl;
#if 0
  err = hipFree(device_sources);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device sources, error: %s\n",
      hipGetErrorString(err));
    throw std::runtime_error("failed to free device sources");
  }
#endif
  return 0;
}

auto makeCompatibleSources(const SourceList& sources) {
  std::vector<SourceCompatibilityData> compat_sources;
  for (const auto& src : sources) {
    compat_sources.push_back(src);
  }
  return compat_sources;
}

static std::vector<std::future<int>> filter_futures_;

void add_filter_future(std::future<int>&& filter_future) {
  filter_futures_.emplace_back(std::move(filter_future));
}

}  // anonymous namespace

namespace cm {

int get_filter_results() {
  for (auto& fut : filter_futures_) {
    if (fut.valid())
      fut.get();
  }
  return 0;
}

void filterCandidatesCuda(
  int sum, int threads_per_block, int num_streams, int stride) {
  add_filter_future(std::async(std::launch::async, filter_task, sum,
    threads_per_block, num_streams, stride));
}

[[nodiscard]] SourceCompatibilityData* cuda_allocCopyXorSources(
  const XorSourceList& xorSourceList) {
  auto xorsrc_bytes = xorSourceList.size() * sizeof(SourceCompatibilityData);
  SourceCompatibilityData* device_xorSources = nullptr;
  hipError_t err = hipMallocAsync(
    (void**)&device_xorSources, xorsrc_bytes, hipStreamPerThread);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device xorSources, error: %s\n",
      hipGetErrorString(err));
    assert(!"failed to allocate device xorSources");
  }
#if 0
  for (size_t i{}; i < xorSourceList.size(); ++i) {
    err = hipMemcpyAsync(&device_xorSources[i], &xorSourceList.at(i),
      sizeof(SourceCompatibilityData), hipMemcpyHostToDevice,
      hipStreamPerThread);
    if (err != hipSuccess) {
      fprintf(stderr, "copy xorSource to device, error: %s\n",
        hipGetErrorString(err));
      assert(!"failed to copy xorSource to device");
    }
  }
#else
  auto compat_sources = makeCompatibleSources(xorSourceList);
  err = hipMemcpyAsync(device_xorSources, compat_sources.data(),
    xorsrc_bytes, hipMemcpyHostToDevice, hipStreamPerThread);
  if (err != hipSuccess) {
    fprintf(
      stderr, "copy xorSource to device, error: %s\n", hipGetErrorString(err));
    assert(!"failed to copy xorSource to device");
  }
#endif
  return device_xorSources;
}

  auto countIndices(const VariationIndicesList& variationIndices) {
    return std::accumulate(variationIndices.cbegin(), variationIndices.cend(),
      0, [](int total, const auto& indices) {
        total += indices.size();
        return total;
      });
  }

  [[nodiscard]] auto cuda_allocCopySentenceVariationIndices(
    const SentenceVariationIndices& sentenceVariationIndices)
    -> device::VariationIndices* {
    hipError_t err = hipSuccess;
    using DeviceVariationIndicesArray =
      std::array<device::VariationIndices, kNumSentences>;
    DeviceVariationIndicesArray deviceVariationIndicesArray;
    for (int s{}; s < kNumSentences; ++s) {
      auto& variationIndices = sentenceVariationIndices.at(s);
      // 2 * size to account for one -1 indices terminator per variation
      const auto device_data_bytes =
        (countIndices(variationIndices) + (2 * variationIndices.size()))
        * sizeof(int);
      auto& deviceVariationIndices = deviceVariationIndicesArray.at(s);
      err = hipMalloc(
        (void**)&deviceVariationIndices.device_data, device_data_bytes);
      assert(err == hipSuccess);
      
      const static int terminator = -1;
      std::vector<int> variationOffsets;
      const auto num_variations{ variationIndices.size() };
      deviceVariationIndices.variationOffsets = deviceVariationIndices.device_data;
      deviceVariationIndices.num_variations = num_variations;
      deviceVariationIndices.sourceIndices =
        &deviceVariationIndices.device_data[num_variations];
      size_t offset{};
      for (const auto& indices: variationIndices) {
        variationOffsets.push_back(offset);
        // NOTE: Async. I'm going to need to preserve sentenceVariationIndices
        // until copy is complete - (kernel execution/synhronize?)
        const auto indices_bytes = indices.size() * sizeof(int);
        err = hipMemcpyAsync(&deviceVariationIndices.sourceIndices[offset],
          indices.data(), indices_bytes, hipMemcpyHostToDevice);
        assert(err == hipSuccess);
        offset += indices.size();
        err = hipMemcpyAsync(&deviceVariationIndices.sourceIndices[offset],
          &terminator, sizeof(terminator), hipMemcpyHostToDevice);
        assert(err == hipSuccess);
        offset += 1;
      }
      const auto variationOffsets_bytes = variationOffsets.size() * sizeof(int);
      err = hipMemcpyAsync(deviceVariationIndices.variationOffsets,
        variationOffsets.data(), variationOffsets_bytes,
        hipMemcpyHostToDevice);
      assert(err == hipSuccess);
    }
    //  const auto sentenceVariationIndices_bytes = 
    //    kNumSentences * sizeof(device::VariationIndices);
    const auto variationIndices_bytes =
      kNumSentences * sizeof(device::VariationIndices);
    device::VariationIndices* device_variationIndices;
    err = hipMalloc((void **)&device_variationIndices, variationIndices_bytes);
    assert(err == hipSuccess);
    
    err = hipMemcpyAsync(device_variationIndices,
                          deviceVariationIndicesArray.data(), variationIndices_bytes,
                          hipMemcpyHostToDevice);
    assert(err == hipSuccess);
    
    return device_variationIndices;
  }

  }  // namespace cm
