#include "hip/hip_runtime.h"
// filter.cu

#include <algorithm>
#include <chrono>
#include <exception>
#include <limits>
#include <numeric>
#include <optional>
#include <thread>
#include <tuple>
#include <hip/hip_runtime.h>
#include "candidates.h"
#include "source-counts.h"

//#define STREAM_LOG

namespace {
  using namespace cm;

/*
__device__ auto isSourceORCompatibleWithAnyOrSource(
  const SourceCompatibilityData& source, const OrSourceList& orSourceList)
{
  auto compatible = false;
  for (const auto& orSource : orSourceList) {
    // skip any sources that were already determined to be XOR incompatible
    // or AND compatible with --xor sources.
    if (!orSource.xorCompatible || orSource.andCompatible) continue;
    compatible = source.isOrCompatibleWith(orSource.source);
    if (compatible) break;
  }
  return compatible;
};

__device__ auto isSourceCompatibleWithEveryOrArg(
  const SourceCompatibilityData& source, const OrArgDataList& orArgDataList)
{
  auto compatible = true; // if no --or sources specified, compatible == true
  for (const auto& orArgData : orArgDataList) {
    // TODO: skip calls to here if container.compatible = true  which may have
    // been determined in Precompute phase @ markAllANDCompatibleOrSources()
    // and skip the XOR check as well in this case.
    compatible = isSourceORCompatibleWithAnyOrSource(source,
     orArgData.orSourceList);
    if (!compatible) break;
  }
  return compatible;
}
  */

  __device__
  auto isSourceXORCompatibleWithAnyXorSource(
    const SourceCompatibilityData& source, const XorSource* xorSources,
    const int* indices, int* compat_index = nullptr, int* reason = nullptr)
  {
    auto compatible{ false }; // important. explicit compatibility required
    for (int i{}; indices[i] > -1; ++i) {
      const auto& xorSource = xorSources[indices[i]];
      compatible = source.isXorCompatibleWith(xorSource, false, reason);
      if (compatible) {
        if (compat_index) *compat_index = indices[i];
        break;
      }
    }
    return compatible;
  }

  __device__ __host__
  auto isSourceXORCompatibleWithAnyXorSource(
    const SourceCompatibilityData& source, const XorSource* xorSources,
    size_t numXorSources, int* compat_index = nullptr, int* reason = nullptr)
  {
    bool compatible = true;
    for (size_t i{}; i < numXorSources; ++i) {
      compatible = source.isXorCompatibleWith(xorSources[i], false, reason);
      if (compatible) {
        if (compat_index) *compat_index = i;
        break;
      }
    }
    return compatible;
  }

  __device__
  auto isSourceXORCompatibleWithAnyXorSource(
    const SourceCompatibilityData& source,
    const XorSource* xorSources, size_t num_xorSources,
    const device::VariationIndices* sentenceVariationIndices,
    int* compat_index = nullptr, int* reason = nullptr)
  {
    for (int s{}; s < kNumSentences; ++s) {
      auto variation = source.usedSources.variations.at(s) + 1;
      const auto& variationIndices = sentenceVariationIndices[s];
      if (!variation || (!variationIndices.num_variations)) {
        continue;
      }
      { // anonymous block
        const auto offset = variationIndices.variationOffsets[variation];
        const auto* indices = &variationIndices.sourceIndices[offset];
        if (isSourceXORCompatibleWithAnyXorSource(source, xorSources,
          indices, compat_index, reason)) return true;
      }
      { // anonymous block
        const auto offset = variationIndices.variationOffsets[0];
        const auto* indices = &variationIndices.sourceIndices[offset];
        if (isSourceXORCompatibleWithAnyXorSource(source, xorSources,
          indices, compat_index, reason)) return true;
      }
      // The idea here is that once we test compatibility with all sources
      // that match the variation of any single sentence of the supplied
      // source, (including with those sources that have no variation for
      // that sentence), we're done. Compatible source(s) were either found
      // for that sentence, or not. We use sentence-compatibility as a shortcut
      // for source-compatibility.
      // TODO: there is an optimization possible here; we could iterate over
      // all sentences first, to identify the sentence with the smallest
      // variation indices list, and test that one, instead of the first one
      // we find.
      return false;
    }
    return isSourceXORCompatibleWithAnyXorSource(source, xorSources,
      num_xorSources, compat_index, reason);
  }

  using result_t = int32_t;

  //__shared__ int compat_index;
  //__shared__ int reason;

  __global__
  void xorKernel(const SourceCompatibilityData* sources, size_t num_sources,
    const XorSource* xorSources, size_t num_xorSources,
    const device::VariationIndices* sentenceVariationIndices,
    const int* source_indices, int stream_index, result_t* results,
    int special)
  {
    //__shared__ SourceCompatibilityData buffer[64];
    constexpr const auto logging = false;

    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_sources) return;
    int compat_index{ -1 }, reason{ -1 };
    auto debug{ true };

    const auto& source = sources[source_indices[index]];
#if 0
    const auto buffer_index = threadIdx.x / blockDim.x +
      threadIdx.x % blockDim.x;
    //for (int i = threadIdx.x / blockDim.x; i < 64; i += warpSize) {
    buffer[buffer_index] = source;
      //}
    //    __syncthreads();
#endif

    const auto* the_source = &source;
    //auto* the_source = &buffer[threadIdx.x];

    bool compat = isSourceXORCompatibleWithAnyXorSource(
      *the_source, xorSources, num_xorSources,
      sentenceVariationIndices, &compat_index,
      debug ? &reason : nullptr);
    if (compat) assert(compat_index > -1); // probably slowish
    results[index] = compat ? compat_index : -1;

    if constexpr (logging) {
      const auto src_index = source_indices[index];
      if (debug) {
        const auto& us = source.usedSources;
        printf("stream %d, index: %d, src_index: %d"
               ", s1 v%d 1st:%d (%d)"
               ", s3 v%d 1st:%d (%d)"
               ", legacy 1st:%d (%d)"
               ", compat: %d, compat_index %d, reason %d\n",
               stream_index, index, src_index,
               us.getVariation(1), us.getFirstSource(1), us.countSources(1),
               us.getVariation(3), us.getFirstSource(3), us.countSources(3),
               source.getFirstLegacySource(), source.countLegacySources(),
               compat, compat_index, reason);
        if (compat) {
          const auto& xor_source = xorSources[compat_index];
          const auto& xor_us = xor_source.usedSources;
          printf("  Xor[%d]: s1 v%d 1st:%d (%d), s3 v%d 1st:%d (%d)"
                 ", legacy 1st:%d (%d)\n",
                 compat_index, xor_us.getVariation(1), xor_us.getFirstSource(1),
                 xor_us.countSources(1), xor_us.getVariation(3),
                 xor_us.getFirstSource(3), xor_us.countSources(3),
                 xor_source.getFirstLegacySource(),
                 xor_source.countLegacySources());
        }
      }
    }
  }

  using ResultList = std::vector<result_t>;

  struct SourceIndex {
    int listIndex{};
    int index{};

    bool operator<(const SourceIndex& rhs) const {
      return (listIndex < rhs.listIndex) || (index < rhs.index);
    }
  };

  struct IndexStates {
    enum class State {
      ready, compatible, done
    };

    struct Data {
      constexpr auto ready_state() const { return state == State::ready; }

      SourceIndex sourceIndex;
      State state = State::ready;
    };
    
    IndexStates() = delete;
    IndexStates(const SourceCompatibilityLists& sources) {
      list.resize(sources.size()); // i.e. "num_sourcelists"
      std::for_each(list.begin(), list.end(), [idx = 0](Data& data) mutable {
        data.sourceIndex.listIndex = idx++;
      });
      for (int flat_index{}; const auto& sourceList: sources) {
        list_sizes.push_back(sourceList.size());
        flat_indices.push_back(flat_index);
        flat_index += sourceList.size();
      }
    }
    
    auto flat_index(SourceIndex src_index) const {
      return flat_indices.at(src_index.listIndex) + src_index.index;
    }
      
    auto list_size(int list_index) const {
      return list_sizes.at(list_index);
    }
      
    auto num_in_state(int first, int count, State state) const {
      int total{};
      for (int i{}; i < count; ++i) {
        if (list.at(first + i).state == state) {
          ++total;
        }
      }
      return total;
    }

    auto num_ready(int first, int count) const {
      return num_in_state(first, count, State::ready);
    }

    auto num_done(int first, int count) const {
      return num_in_state(first, count, State::done);
    }

    auto num_compatible(int first, int count) const {
      return num_in_state(first, count, State::compatible);
    }
 
    auto update(const std::vector<SourceIndex>& sourceIndices,
      ResultList& results, const SourceCompatibilityLists& sources,
      int stream_index) // for logging
    {
      constexpr static const bool logging = false;
      std::set<SourceIndex> compat_src_indices; // for logging
      int num_compatible{};
      int num_done{};
      for (size_t i{}; i < sourceIndices.size(); ++i) {
        const auto result = results.at(i);
        const auto src_index = sourceIndices.at(i);
        auto& indexState = list.at(src_index.listIndex);
        // no longer (always) true, as indexState.index is incremented at
        // fill() time.
        //assert(indexState.sourceIndex.listIndex == src_index.listIndex);
        if constexpr (logging) {
          if (src_index.index == -1) {// && other conditions
            std::cerr << "stream " << stream_index << " "
                      << src_index.listIndex << ":" << src_index.index
                      << ", results index: " << i
                      << ", result: " << result
                      << ", compat: " << std::boolalpha << (result > -1)
                      << ", ready: " << indexState.ready_state()
                      << std::endl;
          }
        }
        // this should only ever happen if the number of lists in "ready"
        // state was less than minimum stride (we doubled up sources from
        // one or more lists).
        if (!indexState.ready_state()) {
          // for debugging purposes, set these "duplicate results for same
          // sentence" results to -1, so we can later determine the exact
          // set of "first matched soruces".
          results.at(i) = -1;
          continue;
        }
        if (result > -1) {
          indexState.state = State::compatible;
          ++num_compatible;
          if constexpr (logging) compat_src_indices.insert(src_index);
        }
        else {
          // if this is the result for the last source in a sourcelist,
          // mark the list (indexState) as done.
          // note that doing it this way *will* put a dependency on the
          // order in which we process sources within a list (currently,
          // in-order), but presumably there'd be some sourceIndexIndices
          // abomination that we could use to determine the *actual* index.
          auto sourcelist_size = (int)sources.at(src_index.listIndex).size();
          if (src_index.index >= sourcelist_size) {
            indexState.state = State::done;
          }
        }
      }
      if constexpr (logging) {
        if (compat_src_indices.size()) {// && (compat_src_indices.size() < 200)) {
          //std::cerr << "stream " << stream_index << " update:";
          for (const auto& src_index: compat_src_indices) {
            std::cout << "" << src_index.listIndex << ":" << src_index.index
                      << std::endl;
          }
          //std::cerr << std::endl;
        }
      }
      return num_compatible;
    }

    auto get(int list_index) const {
      return list.at(list_index);
    }

    auto get_and_increment_index(int list_index) -> std::optional<SourceIndex> {
      auto& data = list.at(list_index);
      if (data.ready_state() &&
         (data.sourceIndex.index < list_sizes.at(list_index)))
      {
        // capture and return value before increment
        auto capture = std::make_optional(data.sourceIndex);
        ++data.sourceIndex.index;
        return capture;
      }
      return std::nullopt;
    }
    
    std::vector<Data> list;
    std::vector<int> flat_indices;
    std::vector<int> list_sizes;
  }; // struct IndexStates

  //////////

  std::vector<hipStream_t> streams;
  int host_special{};
  
  // the pointers in this are allocated in device memory
  struct KernelData {
  private:
    static const auto num_cores = 1280;
    static const auto max_chunks = 20ul;

    static auto calc_stride(const int num_sourcelists) {
      const auto num_chunks = num_sourcelists / max_workitems() + 1;
      assert((num_chunks < max_chunks) && "chunks not supported (but could be)");
      const auto stride = num_sourcelists / num_chunks;
      assert((stride < max_workitems()) && "stride not supported (but could be)");
      return stride;
    }

  public:
    static auto calc_num_streams(const size_t num_sourcelists) {
      const auto num_strides = num_sourcelists / calc_stride(num_sourcelists);
      return std::min(20ul, num_strides + num_strides / 2 + 1);
    }

    static void init(std::vector<KernelData>& kernelVec,
      const size_t num_sourcelists)
    {
      const auto stride = calc_stride(num_sourcelists);
      auto num_primary_streams = num_sourcelists / stride;
      int leftovers = num_sourcelists % stride;
      int start_index{};
      for (size_t i{}; i < kernelVec.size(); ++i) {
        auto& kernel = kernelVec.at(i);
        if (i < num_primary_streams) {
          kernel.list_start_index = start_index;
          int share_of_leftovers = leftovers ? (--leftovers, 1) : 0;
          // this separate accounting of "number of indices" is necessary
          // because source_indices.size() may change, but the number of
          // lists that this stream is concerned with stays constant.
          kernel.num_list_indices = stride + share_of_leftovers;
          start_index += kernel.num_list_indices;
          kernel.source_indices.resize(kernel.num_list_indices);
          
        } else {
          // mark all "extra" streams as unattachle, with no work remaining
          kernel.is_attachable = false;
          kernel.source_indices.resize(0);
        }
        if (i >= streams.size()) {
          hipStream_t stream;
          hipError_t err = hipStreamCreate(&stream);
          assert((err == hipSuccess) && "failed to create stream");
          streams.push_back(stream);
        }
        kernel.stream_index = i;
        kernel.stream = streams[i];
      }
    }

    static int min_workitems(int override = 0) {
      static int the_min_workitems = num_cores + (num_cores / 2);
      if (override) the_min_workitems = override;
      return the_min_workitems;
    }

    static int max_workitems(int override = 0) {
      static int the_max_workitems = 2 * num_cores;
      if (override) the_max_workitems = override;
      return the_max_workitems;
    }

    static int next_sequence_num() {
      static int sequence_num{};
      return sequence_num++;
    }

    //

    int num_ready(const IndexStates& indexStates) const {
      return indexStates.num_ready(list_start_index, num_list_indices);
    }

    int num_done(const IndexStates& indexStates) const {
      return indexStates.num_done(list_start_index, num_list_indices);
    }

    int num_compatible(const IndexStates& indexStates) const {
      return indexStates.num_compatible(list_start_index, num_list_indices);
    }

    auto fillSourceIndices(IndexStates& indexStates, int num_indices) {
      constexpr const auto dupe_checking = false;

      std::set<std::string> dupe_check_indices{}; // debugging
      //std::set<int> list_indices_used{};
      source_indices.resize(num_indices);
      for (int i{}; i < num_indices; /* nothing */) {
        auto any{ false };
        for (int list_offset{}; list_offset < num_list_indices; ++list_offset) {
          const auto list_index = list_start_index + list_offset;
          const auto opt_src_index =
            indexStates.get_and_increment_index(list_index);
          if (opt_src_index.has_value()) {
            const auto src_index = opt_src_index.value();
            assert(src_index.listIndex == list_index);

            // DEBUGGGGGGGGGG
            #if 0
            if (src_index.listIndex == -1) {
              host_special = i;
            }
            if (src_index.listIndex == -1) {
              const auto flat_index = indexStates.flat_index(src_index);
              //const auto list_size = indexStates.flat_index(src_index.listIndex + 1) -
              //indexStates.flat_index(src_index.listIndex);
              std::cerr << "flat_index: " << flat_index << " = "
                        << src_index.listIndex << ":" << src_index.index
                        << ", list size: " << indexStates.list_size(src_index.listIndex)
                        << std::endl;
            }
            #endif

            source_indices.at(i++) = src_index;
            // TODO this is slow and only used for logging and I don't like it.
            // figure it out.
            //list_indices_used.insert(src_index.listIndex);

            if constexpr (dupe_checking) {
              char buf[32];
              snprintf(buf, sizeof(buf), "%d:%d", src_index.listIndex,
                src_index.index);
              std::string str_index{ buf };
              if (!dupe_check_indices.insert(str_index).second) {
                std::cerr << "stream " << stream_index << ": duplicate index: "
                          << str_index << std::endl;
              }
            }

            any = true;
            if (i >= num_indices) break;
          }
        }
        if (!any) {
          source_indices.resize(i);
          break;
        }
      }
      return 8008135; // list_indices_used.size();
    }

    bool fillSourceIndices(IndexStates& indexStates) {
      constexpr static const auto logging = false;
      auto num_ready = indexStates.num_ready(list_start_index,
        num_list_indices);
      int num_sourcelists{};
      if (num_ready) {
        auto num_indices = num_ready;
        if (num_ready < num_list_indices) num_indices = max_workitems();
        if (is_attached()) num_indices = min_workitems();
        // TODO: should probably be a percentage, not a fixed #
        //if (num_ready < 250) num_indices = max_workitems();
        num_sourcelists = fillSourceIndices(indexStates, num_indices);
      } else {
        source_indices.resize(0);
      }
      if (source_indices.empty()) {
        if constexpr (logging) {
          std::cerr << "  fill " << stream_index << ": empty " << std::endl;
        }
        return false;
      }
      if constexpr (logging) {
        std::cerr << "  fill " << stream_index << ":"
                  << " added " << source_indices.size() << " sources"
                  << " from " << num_sourcelists << " sourcelists"
                  << " (" << list_start_index << " - "
                  << list_start_index + num_list_indices - 1 << ")"
                  << std::endl;
      }
      return true;
    }

    void allocCopy(const IndexStates& indexStates) {
      hipError_t err = hipSuccess;
      // alloc source indices
      if (!device_source_indices) {
        auto sources_bytes = max_workitems() * sizeof(int);
        err = hipMallocAsync((void **)&device_source_indices, sources_bytes,
          stream);
        assert((err == hipSuccess) && "failed to allocate source indices");
      }

      std::vector<int> flat_indices;
      flat_indices.reserve(source_indices.size());
      for (const auto& src_index: source_indices) {
        flat_indices.push_back(indexStates.flat_index(src_index));
      }
      // copy (flat) source indices
      #if 0
      std::cerr << "copying " << flat_indices.size() << " indices"
                << " to device_indices(" << device_source_indices << ")"
                << " on stream " << stream_index << " (" << stream << ")"
                << std::endl;
      #endif
      err = hipMemcpyAsync(device_source_indices, flat_indices.data(),
        flat_indices.size() * sizeof(int), hipMemcpyHostToDevice, stream);
      assert((err == hipSuccess) && "failed to copy source indices");
      
      // alloc results
      if (!device_results) {
        auto results_bytes = max_workitems() * sizeof(result_t);
        err = hipMallocAsync((void **)&device_results, results_bytes, stream);
        if (err != hipSuccess) {
          fprintf(stderr, "Failed to allocate stream %d results, error: %s\n",
            stream_index, hipGetErrorString(err));
          throw std::runtime_error("failed to allocate results");
        }
      }
      /*
      constexpr static bool debug_indices = true;
      if constexpr (debug_indices) {
        // alloc indices (debugging)
        if (!device_compat_indices) {
          auto indices_bytes = max_workitems() * sizeof(int);
          err = hipMallocAsync((void **)&device_compat_indices, indices_bytes, stream);
          if (err != hipSuccess) {
            fprintf(stderr, "Failed to allocate stream %d indices, error: %s\n",
              stream_index, hipGetErrorString(err));
            throw std::runtime_error("failed to allocate results");
          }
        }
      }
      */
    }

    auto hasWorkRemaining() const {
      return !source_indices.empty();
    }

    void attach(struct KernelData& kernel) {
      assert(!is_running && !is_attached() && kernel.is_running);
      num_attached = 0;
      attached_to = kernel.stream_index;
      is_attachable = false;
      has_run = true;
      list_start_index = kernel.list_start_index;
      num_list_indices = kernel.num_list_indices;
      // signal work remaining
      source_indices.resize(1);

      kernel.num_attached++;

#ifdef STREAM_LOG
      std::cerr << "stream " << stream_index << " attaching to stream "
                << kernel.stream_index << std::endl;
#endif
    }

    bool is_attached() const { return attached_to > -1; }

    int list_start_index; // starting index in SourceCompatibiliityLists
    int num_list_indices; // # of above list entries we are concerned with
    int stream_index{ -1 };
    int num_attached{};
    int sequence_num{};
    int attached_to{ -1 };      // stream_index of stream we're attached to
    bool is_attachable{ true }; // can be attached to 
    bool is_running{ false };   // is running (may be complete; output not retrieved)
    bool has_run{ false };      // has run at least once
    int* device_source_indices{ nullptr }; // in
    result_t *device_results{ nullptr }; // out
    hipStream_t stream{ nullptr };
    std::vector<SourceIndex> source_indices;  // .size() == num_results
    //result_t *device_compat_indices{ nullptr }; 
  }; // struct KernelData

  //////////

  struct ValueIndex {
    int value{};
    int index{ -1 };
  };

  auto anyWithWorkRemaining(const std::vector<KernelData>& kernelVec,
    bool attachable_only = false) -> std::optional<int>
  {
    // NOTE: list initialization
    ValueIndex fewest_attached = { std::numeric_limits<int>::max() };
    for (size_t i{}; i < kernelVec.size(); ++i) {
      const auto& kernel = kernelVec[i];
      if (kernel.hasWorkRemaining() &&
        (!attachable_only || kernel.is_attachable))
      {
        if (kernel.num_attached < fewest_attached.value) {
          fewest_attached.value = kernel.num_attached;
          fewest_attached.index = i;
          if (!fewest_attached.value) break;
        }
      }
    }
    if (fewest_attached.index > -1) {
      return std::make_optional(fewest_attached.index);
    }
    return std::nullopt;
  }

  bool anyIdleWithWorkRemaining(const std::vector<KernelData>& kernelVec,
    int& index)
  {
    for (size_t i{}; i < kernelVec.size(); ++i) {
      const auto& kernel = kernelVec[i];
      if (!kernel.is_running && kernel.hasWorkRemaining()) {
        index = i;
        return true;
      }
    }
    return false;
  }

  auto anyReadyToAttach(const std::vector<KernelData>& kernelVec)
    -> std::optional<int>
  {
    for (size_t i{}; i < kernelVec.size(); ++i) {
      const auto& kernel = kernelVec[i];
      if (!kernel.is_running && !kernel.is_attached() &&
          !kernel.hasWorkRemaining())
      {
        return std::make_optional(i);
      }
    }
    return std::nullopt;
  }

  // TODO: std::optional, and above here
  bool anyRunningComplete(const std::vector<KernelData>& kernelVec,
    int& index)
  {
    // NOTE: list initialization
    ValueIndex lowest = { std::numeric_limits<int>::max() };
    for (size_t i{}; i < kernelVec.size(); ++i) {
      const auto& kernel = kernelVec[i];
      if (kernel.is_running && (hipSuccess == hipStreamQuery(kernel.stream))) {
        if (kernel.sequence_num < lowest.value) {
          lowest.value = kernel.sequence_num;
          lowest.index = i;
        }
      }
    }
    if (lowest.index > -1) {
      index = lowest.index;
      return true;
    }
    return false;
  }

  bool getNextWithWorkRemaining(std::vector<KernelData>& kernelVec,
    int& current)
  {
    using namespace std::chrono_literals;

    // First: ensure all primary streams have started at least once
    if (++current >= (int)kernelVec.size()) {
      current = 0;
    } else {
      const auto& kernel = kernelVec[current];
      if (!kernel.is_running && !kernel.has_run && kernel.hasWorkRemaining()) {
        return true;
      }
    }

    // Second: process results for any "running" stream that has completed
    if (anyRunningComplete(kernelVec, current)) {
      return true;
    }

    // Third: run any idle (non-running) stream with work remaining
    if (anyIdleWithWorkRemaining(kernelVec, current)) {
      return true;
    }

    // There is no idle stream that has work remaining. Is there an attachable
    // running stream that has work remaining?
    std::optional<int> opt_attachable = anyWithWorkRemaining(kernelVec, true);
    if (opt_attachable.has_value()) {
      // Is there an idle stream that can attach to it? If so, attach it.
      std::optional<int> opt_attach = anyReadyToAttach(kernelVec);
      if (opt_attach.has_value()) {
        kernelVec.at(opt_attach.value())
          .attach(kernelVec.at(opt_attachable.value()));
        current = opt_attach.value();
        return true;
      }
    } else {
      // There is no idle stream, and no attachable running stream that has work
      // remaining. Is there any stream with work remaining? If not, we're done.
      if (!anyWithWorkRemaining(kernelVec).has_value()) {
        return false;
      }
    }

    // Wait for one to complete.
    while (!anyRunningComplete(kernelVec, current)) {
      // TODO events
      std::this_thread::sleep_for(10ms);
      //std::this_thread::yield();
    }
    return true;
  }

  void runKernel(KernelData& kernel,
    const SourceCompatibilityData* device_sources)
  {
    auto num_sources = kernel.source_indices.size();
    int threadsPerBlock = 32;
    int blocksPerGrid = (num_sources + threadsPerBlock - 1) / threadsPerBlock;
    kernel.is_running = true;
    kernel.sequence_num = KernelData::next_sequence_num();
    xorKernel<<<blocksPerGrid, threadsPerBlock, 0, kernel.stream>>>(
      device_sources, num_sources,
      PCD.device_xorSources, PCD.xorSourceList.size(),
      PCD.device_sentenceVariationIndices, kernel.device_source_indices,
      kernel.stream_index, kernel.device_results, host_special);

#ifdef STREAM_LOG
    std::cerr << "  stream " << kernel.stream_index << " starting, sequence: "
              << kernel.sequence_num << std::endl;
#endif
    #if 0 || defined(DEBUG)
    fprintf(stderr, "  stream %d (%d) launched with %d blocks of %d threads\n",
      kernel.stream_index, kernel.sequence_num, blocksPerGrid, threadsPerBlock);
    #endif
  }

  // todo: kernel.getResults()
  auto getKernelResults(KernelData& kernel) {
    auto num_sources = kernel.source_indices.size();
    ResultList results(num_sources);
    auto results_bytes = num_sources * sizeof(result_t);
    hipStreamSynchronize(kernel.stream);
    hipError_t err = hipMemcpyAsync(results.data(), kernel.device_results,
      results_bytes, hipMemcpyDeviceToHost, kernel.stream);
    assert((err == hipSuccess) && "failed to copy results from device -> host");
    return results;
  }

  void showAllNumReady(const std::vector<KernelData>& kernels,
    const IndexStates& indexStates)
  {
    for (auto& k: kernels) {
      std::cerr << "  stream " << k.stream_index << ": " 
                << k.num_ready(indexStates) << std::endl;
    }
  }

  auto count(const SourceCompatibilityLists& sources) {
    size_t num{};
    for (const auto& sourceList: sources) {
      num += sourceList.size();
    }
    return num;
  }

  auto* allocCopySources(const SourceCompatibilityLists& sources) {
    // alloc sources
    hipError_t err = hipSuccess;
    auto sources_bytes = count(sources) * sizeof(SourceCompatibilityData);
    SourceCompatibilityData* device_sources;
    err = hipMalloc((void **)&device_sources, sources_bytes);
    assert((err == hipSuccess) && "failed to allocate sources");

    // copy sources
    size_t index{};
    for (const auto& sourceList: sources) {
      auto sourceIndices = cm::getSortedSourceIndices(sourceList, false);
      if (sourceIndices.size()) {
        for (size_t i{}; i < sourceIndices.size(); ++i) {
          const auto& src = sourceList.at(sourceIndices.at(i));
          err = hipMemcpy(&device_sources[index++], &src,
            sizeof(SourceCompatibilityData), hipMemcpyHostToDevice);
          assert((err == hipSuccess) && "failed to copy source");
        }
      } else {
        err = hipMemcpy(&device_sources[index], sourceList.data(),
          sourceList.size() * sizeof(SourceCompatibilityData),
          hipMemcpyHostToDevice);
        assert((err == hipSuccess) && "failed to copy sources");
        index += sourceList.size();
      }
    }
    return device_sources;
  }

  void printCompatRecord(const std::vector<std::vector<int>>& compat_record) {
    int total{};
    for (size_t i{}; i < compat_record.size(); ++i) {
      const auto& counts = compat_record.at(i);
      std::cerr << "stream " << i << ":";
      for (auto count: counts) {
        std::cerr << " " << count;
        total += count;
      }
      std::cerr << std::endl;
    }
    std::cerr << "total: " << total << std::endl;
  }

  int median(std::vector<int>& a) {
    const int n = a.size();
    if (!(n % 2)) {
      std::nth_element(a.begin(), a.begin() + n / 2, a.end());
      std::nth_element(a.begin(), a.begin() + (n - 1) / 2, a.end());
      return (a[(n - 1) / 2] + a[n / 2]) / 2;
    }
    std::nth_element(a.begin(), a.begin() + n / 2, a.end());
    return a[n / 2];
  }

  auto sources_stats(const SourceCompatibilityLists& sources) {
    std::vector<int> sizes;
    sizes.reserve(sources.size());
    size_t mode{}, sum{};
    for (const auto& src_list: sources) {
      const auto size = src_list.size();
      if (size > mode) {
        mode = size;
      }
      sum += size;
      sizes.push_back(size);
    }
    // mean, median, mode
    return std::make_tuple(sum / sources.size(), median(sizes), mode);
  }

  void check(const SourceCompatibilityLists& sources, int list_index, int index)
  {
    constexpr const auto logging = true;
    if constexpr (logging) {
      char buf[32];
      snprintf(buf, sizeof(buf), "%d:%d", list_index, index);
      auto& source = sources.at(list_index).at(index);
      source.dump(buf);
      int compat_index{ -1 };
      auto compat = isSourceXORCompatibleWithAnyXorSource(source,
        PCD.xorSourceList.data(), PCD.xorSourceList.size(), &compat_index);
      std::cerr << "compat: " << compat << " (" << compat_index << ")"
                << std::endl;
    }
  }

  void dump(const XorSourceList& xorSources,
    const std::vector<int>& xorSourceIndices, int index)
  {
    auto host_index = xorSourceIndices.at(index);
    const auto& src = xorSources.at(host_index);
    char buf[32];
    snprintf(buf, sizeof(buf), "xor: device(%d) host(%d)", index, host_index);
    src.dump(buf);
  }
} // anonymous namespace

namespace cm {

void filterCandidatesCuda(int sum, int num_streams, int workitems) {
  using namespace std::chrono;

  const auto& sources = allSumsCandidateData.find(sum)->second
    .sourceCompatLists;
  auto device_sources = allocCopySources(sources);

  const auto [mean, median, mode] = sources_stats(sources);
  const auto src_data_bytes = sizeof(SourceCompatibilityData);
  const auto necessary_bytes = sizeof(LegacySources) + sizeof(Sources) +
    sizeof(UsedSources::Variations);
  std::cerr << "sizeof SourceData: " << src_data_bytes
            << ", necessary: " << necessary_bytes
            << ", mean: " << mean
            << " (" << mean * src_data_bytes
            << ", " << mean * necessary_bytes << ")"
            << ", median: " << median
            << " (" << median * src_data_bytes
            << ", " << median * necessary_bytes << ")"
            << ", mode: " << mode
            << " (" << mode * src_data_bytes
            << ", " << mode * necessary_bytes << ")"
            << std::endl;

  auto t0 = high_resolution_clock::now();
  //KernelData::min_workitems(workitems);
  KernelData::max_workitems(workitems);
  if (!num_streams) num_streams = KernelData::calc_num_streams(sources.size());
  std::vector<KernelData> kernels(num_streams);
  KernelData::init(kernels, sources.size());
  std::cerr << "sourcelists: " << sources.size()
            << ", workitems: " << KernelData::max_workitems()
            << ", streams: " << num_streams
            << ", primary: " << sources.size() / KernelData::max_workitems()
            << std::endl;

  std::set<int> compat_indices;
  std::vector<std::vector<int>> compat_record(num_streams);
  IndexStates indexStates{ sources };
  int total_compatible{};
  int current_kernel{ -1 };
  while (getNextWithWorkRemaining(kernels, current_kernel)) {
    auto& kernel = kernels.at(current_kernel);
    if (!kernel.is_running) {
      if (kernel.fillSourceIndices(indexStates)) {
        // TODO: move alloc to separate func outside loop
        // consider copying all source data on stream0, 
        // and only copy indices array here
        kernel.allocCopy(indexStates);
        runKernel(kernel, device_sources);
      } else if (kernel.is_attached()) {
#ifdef STREAM_LOG
        std::cerr << "stream " << kernel.stream_index
                  << " detached from " << kernel.attached_to
                  << " - no more work"
                  << std::endl;
#endif
        kernels.at(kernel.attached_to).is_attachable = false;
        kernel.attached_to = -1;
      }
      continue;
    }

    kernel.has_run = true;
    kernel.is_running = false;

    auto r0 = high_resolution_clock::now();
    auto results = getKernelResults(kernel);
    auto r1 = high_resolution_clock::now();
    auto d_results = duration_cast<milliseconds>(r1 - r0).count();

    auto num_compatible = indexStates.update(kernel.source_indices,
      results, sources, kernel.stream_index);
    total_compatible += num_compatible;

#if 0
    for (size_t r{}; r < kernel.source_indices.size(); ++r) {
      const auto result = results.at(r);
      if (result > -1) compat_indices.insert(result);
    }
    compat_record.at(kernel.stream_index).push_back(num_compatible);
#endif

#ifdef STREAM_LOG
    std::cerr << "  stream " << current_kernel << " done"
      //<< ", done: " << kernel.num_done(indexStates)
      //<< ", compat reported: " << num_compatible
      //<< ", compat actual:" << kernel.num_compatible(indexStates)
      //<< ", total compatible: " << total_compatible
      // TODO remaining is showing weird value, look into it
      //<< ", remaining: " << kernel.num_ready(indexStates)
              << " - " << d_results << "ms" << std::endl;
#endif
    #ifdef DEBUG
    assert(kernel.num_list_indices == kernel.num_ready(indexStates) +
      kernel.num_compatible(indexStates) + kernel.num_done(indexStates));
    #endif
  }
  auto t1 = high_resolution_clock::now();
  auto d_kernel = duration_cast<milliseconds>(t1 - t0).count();

  //printCompatRecord(compat_record);
  std::cerr << "total compatible: " << total_compatible << " of "
            << sources.size() << " - " << d_kernel << "ms"
            << std::endl;
  #if 0
  for (auto index: compat_indices) {
    std::cout << index << std::endl;
  }
  #endif

  /*
  //  auto num_sources = count(compatLists);
  auto source_bytes = num_sources * sizeof(SourceCompatibilityData);

  auto ac0 = high_resolution_clock::now();
  // begin alloc_copy 

  // end alloc-copy
  auto ac1 = high_resolution_clock::now();
  auto dur_ac = duration_cast<milliseconds>(ac1 - ac0).count();
  std::cerr << "  alloc/copy " << compatLists.size() << " compatLists"
            << " (" << num_sources << ") done - " << dur_ac << "ms"
            << std::endl;
  */

//#define IMMEDIATE_RESULTS
#ifdef IMMEDIATE_RESULTS
  std::vector<result_t> results;
  results.resize(num_source);
  err = hipMemcpy(results.data(), device_results, results_bytes,
                   hipMemcpyDeviceToHost, stream);

  auto& indexComboListMap = allSumsCandidateData.find(sum)->second
    .indexComboListMap;
  int num_compat_combos{};
  int num_compat_sourcelists{};
  index = 0;
  int list_index{};
  for (const auto& compatList: compatLists) {
    int result_index{ index };
    for (const auto& source: compatList) {
      if (results.at(result_index)) {
        ++num_compat_sourcelists;
        num_compat_combos += indexComboListMap.at(list_index).size();
        break;
      }
      result_index++;
    }
    index += compatList.size();
    ++list_index;
  }
  int num_compat_results = std::accumulate(results.cbegin(), results.cend(), 0,
    [](int num_compatible, result_t result) mutable {
      if (result) num_compatible++;
      return num_compatible;
    });
  std::cerr << "  results: " << results.size()
    << ", compat results: " << num_compat_results
    << ", compat sourcelists: " << num_compat_sourcelists
    << ", compat combos: " << num_compat_combos
    << std::endl;

  err = hipFree(device_results);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device results (error code %s)!\n",
      hipGetErrorString(err));
    throw std::runtime_error("failed to free device results");
  }
#endif // IMMEDIATE_RESULTS

#if 0
  err = hipFree(device_sources);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device sources, error: %s\n",
      hipGetErrorString(err));
    throw std::runtime_error("failed to free device sources");
  }
#endif
}

[[nodiscard]]
XorSource* cuda_allocCopyXorSources(const XorSourceList& xorSourceList,
  const std::vector<int> sortedIndices)
{
  auto xorSources_bytes = xorSourceList.size() * sizeof(XorSource);
  XorSource *device_xorSources = nullptr;
  hipError_t err = hipMalloc((void **)&device_xorSources, xorSources_bytes);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device xorSources, error: %s\n",
      hipGetErrorString(err));
    throw std::runtime_error("failed to allocate device xorSources");
  }
  for (size_t i{}; i < sortedIndices.size(); ++i) {
    err = hipMemcpyAsync(&device_xorSources[i],
      &xorSourceList.at(sortedIndices[i]), sizeof(XorSource),
      hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to copy xorSources host -> device, error: %s\n",
              hipGetErrorString(err));
      throw std::runtime_error("failed to copy xorSources host -> device");
    }
  }
  return device_xorSources;
}

auto countIndices(const VariationIndicesList& variationIndices) {
  return std::accumulate(variationIndices.cbegin(), variationIndices.cend(), 0,
    [](int total, const auto& indices) {
      total += indices.size();
      return total;
    });
}

[[nodiscard]]
auto cuda_allocCopySentenceVariationIndices(
  const SentenceVariationIndices& sentenceVariationIndices)
  -> device::VariationIndices*
{
  hipError_t err = hipSuccess;
  using DeviceVariationIndicesArray =
    std::array<device::VariationIndices, kNumSentences>;
  DeviceVariationIndicesArray deviceVariationIndicesArray;
  for (int s{}; s < kNumSentences; ++s) {
    auto& variationIndices = sentenceVariationIndices.at(s);
    // 2 * size to account for one -1 indices terminator per variation
    const auto device_data_bytes = (countIndices(variationIndices) +
      (2 * variationIndices.size())) * sizeof(int);
    auto& deviceVariationIndices = deviceVariationIndicesArray.at(s);
    err = hipMalloc((void **)&deviceVariationIndices.device_data,
      device_data_bytes);
    assert(err == hipSuccess);

    const static int terminator = -1;
    std::vector<int> variationOffsets;
    const auto num_variations{ variationIndices.size() };
    deviceVariationIndices.variationOffsets = deviceVariationIndices.device_data;
    deviceVariationIndices.num_variations = num_variations;
    deviceVariationIndices.sourceIndices =
      &deviceVariationIndices.device_data[num_variations];
    size_t offset{};
    for (const auto& indices: variationIndices) {
      variationOffsets.push_back(offset);
      // NOTE: Async. I'm going to need to preserve sentenceVariationIndices
      // until copy is complete - (kernel execution/synhronize?)
      const auto indices_bytes = indices.size() * sizeof(int);
      err = hipMemcpyAsync(&deviceVariationIndices.sourceIndices[offset],
        indices.data(), indices_bytes, hipMemcpyHostToDevice);
      assert(err == hipSuccess);
      offset += indices.size();
      err = hipMemcpyAsync(&deviceVariationIndices.sourceIndices[offset],
        &terminator, sizeof(terminator), hipMemcpyHostToDevice);
      assert(err == hipSuccess);
      offset += 1;
    }
    const auto variationOffsets_bytes = variationOffsets.size() * sizeof(int);
    err = hipMemcpyAsync(deviceVariationIndices.variationOffsets,
      variationOffsets.data(), variationOffsets_bytes, hipMemcpyHostToDevice);
    assert(err == hipSuccess);
  }
  //  const auto sentenceVariationIndices_bytes = 
  //    kNumSentences * sizeof(device::VariationIndices);
  const auto variationIndices_bytes =
    kNumSentences * sizeof(device::VariationIndices);
  device::VariationIndices* device_variationIndices;
  err = hipMalloc((void **)&device_variationIndices, variationIndices_bytes);
  assert(err == hipSuccess);

  err = hipMemcpyAsync(device_variationIndices,
    deviceVariationIndicesArray.data(), variationIndices_bytes,
    hipMemcpyHostToDevice);
  assert(err == hipSuccess);

  return device_variationIndices;
}

} // namespace cm
