#include "hip/hip_runtime.h"
// filter.cu

#include <algorithm>
#include <chrono>
#include <exception>
#include <numeric>
#include <hip/hip_runtime.h>
#include "candidates.h"

namespace {
  using namespace cm;

  /*
__device__ auto isSourceORCompatibleWithAnyOrSource(
  const SourceCompatibilityData& source, const OrSourceList& orSourceList)
{
  auto compatible = false;
  for (const auto& orSource : orSourceList) {
    // skip any sources that were already determined to be XOR incompatible
    // or AND compatible with --xor sources.
    if (!orSource.xorCompatible || orSource.andCompatible) continue;
    compatible = source.isOrCompatibleWith(orSource.source);
    if (compatible) break;
  }
  return compatible;
};

__device__ auto isSourceCompatibleWithEveryOrArg(
  const SourceCompatibilityData& source, const OrArgDataList& orArgDataList)
{
  auto compatible = true; // if no --or sources specified, compatible == true
  for (const auto& orArgData : orArgDataList) {
    // TODO: skip calls to here if container.compatible = true  which may have
    // been determined in Precompute phase @ markAllANDCompatibleOrSources()
    // and skip the XOR check as well in this case.
    compatible = isSourceORCompatibleWithAnyOrSource(source,
     orArgData.orSourceList);
    if (!compatible) break;
  }
  return compatible;
}
  */

  /*
//__device__
auto isSourceXORCompatibleWithAnyXorSource(
  const SourceCompatibilityData& source, const XorSourceList& xorSourceList,
  const std::vector<int>& indices)
{
  bool compatible = true; // empty list == compatible
  for (auto index : indices) {
    const auto& xorSource = xorSourceList[index];
    compatible = source.isXorCompatibleWith(xorSource, false);
    if (compatible) break;
  }
  return compatible;
};

//__device__
auto isSourceXORCompatibleWithAnyXorSource(
  const SourceCompatibilityData& source, const XorSourceList& xorSourceList,
  const std::array<VariationIndicesMap, kNumSentences>& variationIndicesMaps)
{
  for (auto s = 0; s < kNumSentences; ++s) {
    auto variation = source.usedSources.variations[s];
    const auto& map = variationIndicesMaps[s];
    if ((variation < 0) || (map.size() == 1)) continue;
    if (auto it = map.find(variation); it != map.end()) {
      if (isSourceXORCompatibleWithAnyXorSource(source, xorSourceList,
        it->second))
      {
        return true;
      }
    }
    if (auto it = map.find(-1); it != map.end()) {
      if (isSourceXORCompatibleWithAnyXorSource(source, xorSourceList,
        it->second))
      {
        return true;
      }
    }
    return false;
  }
  return isSourceXORCompatibleWithAnyXorSource(source, xorSourceList,
    variationIndicesMaps[1].at(-1)); // hack: we know "sentence" 2 doesn't exist
}
  */

  /*
__host__ __device__ bool isAnySourceCompatibleWithUseSources(
  const SourceCompatibilityData *source, int numCompatData,
  const XorSource* xorSources, size_t numXorSources)
{
  if (!numCompatData) return true;
  auto compatible = false;
  for (auto i = 0; i < numCompatData; ++i) {
    compatible = isSourceXORCompatibleWithAnyXorSource(source[i],
      xorSources, numXorSources); // , PCD.variationIndicesMaps);
    // if there were --xor sources specified, and none are compatible with the
    // current source, no further compatibility checking is necessary; continue
    // to next source.
    if (!compatible) continue;
    // TODO
    //compatible = isSourceCompatibleWithEveryOrArg(source[i],
    //  PCD.orArgDataList);
    if (compatible) break;
  }
  return compatible;
};
  */

  /*
__host__ __device__
void strcat_char(char* buf, char c) {
  while (*buf) buf++;
  *(buf++) = c;
  *buf = 0;
}

__host__ __device__
void strcat_int(char* buf, int i) {
  while (*buf) buf++;
  int factor = 100;
  while (factor) {
    int val = i / factor;
    if (val > 0) {
      *(buf++) = val + '0';
      i %= factor;
    }
    factor /= 10;
  }
  *buf = 0;
}

__host__ __device__
char* buildLegacySourcesString(const SourceCompatibilityData& scd, char* buf){
  *buf = 0;
  for (int i{}; i < kMaxLegacySources; ++i) {
    if (scd.legacySources[i]) {
      strcat_char(buf, ' ');
      strcat_int(buf, i);
    }
  }
  return buf;
}

__host__ __device__
char* buildSourcesString(const SourceCompatibilityData& scd, char* buf){
  *buf = 0;
  for (int s{1}; s <= kNumSentences; ++s) {
    auto first = Source::getFirstIndex(s);
    for (int i{}; i < kMaxUsedSourcesPerSentence; ++i) {
      if (scd.usedSources.sources[first + i] == -1) break;
      strcat_char(buf, ' ');
      strcat_int(buf, s);
      strcat_char(buf, ':');
      strcat_int(buf, scd.usedSources.sources[first + i]);
    }
  }
  return buf;
}

__host__ __device__ void printSources(const SourceCompatibilityData& scd) {
  char buf[256];
  buildLegacySourcesString(scd, buf);
  printf("  legacySources %s\n", buf);
  buildSourcesString(scd, buf);
  printf("  sources %s\n", buf);
}
  */
  
  __host__ __device__ auto isSourceXORCompatibleWithAnyXorSource(
    const SourceCompatibilityData& source,
    const XorSource* xorSources, size_t numXorSources,
    int* outIndex = nullptr)
  {
    bool compatible = true; // empty list == compatible
    for (auto i = 0u; i < numXorSources; ++i) {
      compatible = source.isXorCompatibleWith(xorSources[i], false);
      if (compatible) {
        if (outIndex) *outIndex = i;
        break;
      }
    }
    return compatible;
  }

  using result_t = uint8_t;

  __global__ void kernel(const SourceCompatibilityData* sources, size_t num_sources, 
    const XorSource* xorSources, size_t num_xorSources, result_t* results)
  {
    auto index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_sources) return;
    bool compat = isSourceXORCompatibleWithAnyXorSource(sources[index],
      xorSources, num_xorSources);
    results[index] = compat ? 1 : 0;
  }

  /*
  auto count(const SourceCompatibilityLists& compatLists) {
    size_t num{};
    for (const auto& compatList: compatLists) {
      for (auto& source: compatList) {
        num++;
      }
    }
    return num;
  }
  */

  using ResultList = std::vector<result_t>;

  struct SourceIndex {
      int listIndex{};
      int index{};
  };

  struct IndexStates {
    enum class State {
      ready, compatible, done
    };

    struct Data {
      constexpr auto ready_state() const { return state == State::ready; }

      SourceIndex sourceIndex;
      State state = State::ready;
    };
    
    IndexStates() = delete;
    IndexStates(size_t size) {
      list.resize(size);
      std::for_each(list.begin(), list.end(), [idx = 0](Data& data) mutable {
        data.sourceIndex.listIndex = idx++;
      });
    }

    auto num_in_state(int first, int count, State state) const {
      int total{};
      for (int i{}; i < count; ++i) {
        if (list.at(first + i).state == state) total++;
      }
      return total;
    }

    auto num_ready(int first, int count) const {
      return num_in_state(first, count, State::ready);
    }

    auto num_done(int first, int count) const {
      return num_in_state(first, count, State::done);
    }

    auto num_compatible(int first, int count) const {
      return num_in_state(first, count, State::compatible);
    }

    /*
    auto get_flat_index(SourceIndex sourceIndex, 
      const SourceCompatibilityLists& sources)
    {
      int flat_index{ sourceIndex.index };
      for (int i{}; i < sourceIndex.listIndex; ++i) {
        flat_index += sources[i].size();
      }
      return flat_index;
    }
    */

    auto update(const std::vector<SourceIndex>& sourceIndices,
      const ResultList& results, const SourceCompatibilityLists& sources)
    {
      int num_compatible{};
      for (int i{}; i < sourceIndices.size(); ++i) {
        const auto result = results.at(i);
        assert(result == 0 || result == 1); // sanity check
        auto& data = list.at(sourceIndices.at(i).listIndex);
        assert(data.sourceIndex.listIndex == sourceIndices.at(i).listIndex);
        // this should only ever happen if the number of lists in "ready"
        // state was less than minimum stride (we doubled up sources from
        // one or more lists).
        if (!data.ready_state()) continue;
        if (result) {
          data.state = State::compatible;
          num_compatible++;
        } else {
          auto sourcelist_size = sources.at(data.sourceIndex.listIndex).size();
          if (++data.sourceIndex.index >= sourcelist_size) {
            data.state = State::done;
          }
        }
      }
      return num_compatible;
    }
    
    std::vector<Data> list;
  }; // struct IndexStates

  //////////

  std::vector<hipStream_t> streams;
  
  // the pointers in this are allocated in device memory
  struct KernelData {
  private:
    static const int magic_multiple = 2;
    static const int num_cores = 1280;
    static const int min_workitems = magic_multiple * num_cores;

    /*
    static int next_stream_index() {
      static int next = 0;
      return next++;
    }
    */

  public:
    //KernelData(): stream_index(next_stream_index()) {}

    //constexpr
    static int getNumStreams(size_t num_sources) {
      return std::min(24ul, num_sources / min_workitems + 1);
    }

    static void init(std::vector<KernelData>& dataVec, int num_sources) {
      const int stride = num_sources / dataVec.size();
      int start_index{};
      for (auto i{ 0u }; i < dataVec.size(); ++i) {
        auto& data = dataVec.at(i);
        data.list_start_index = start_index;
        int remain = num_sources - start_index;
        data.source_indices.resize(remain < stride ? remain : stride);
        // this is necessary because source_indices.size() may change, but the
        // number of list_indices this kernel is concerned with remains constant
        data.num_list_indices = data.source_indices.size();
        start_index += stride;
        if (i >= streams.size()) {
          hipStream_t stream;
          hipError_t err = hipStreamCreate(&stream);
          assert(err == hipSuccess);
          streams.push_back(stream);
        }
        data.stream_index = i;
        data.stream = streams[i];
      }
    }

    //

    int num_ready(const IndexStates& indexStates) const {
      return indexStates.num_ready(list_start_index, num_list_indices);
    }

    int num_done(const IndexStates& indexStates) const {
      return indexStates.num_done(list_start_index, num_list_indices);
    }

    int num_compatible(const IndexStates& indexStates) const {
      return indexStates.num_compatible(list_start_index, num_list_indices);
    }

    bool fillSourceIndices(const IndexStates& indexStates) {
      auto num_ready = indexStates.num_ready(list_start_index, num_list_indices);
      std::set<int> list_indices{}; // logging
      if (num_ready) {
        auto num_indices = num_ready < min_workitems ? min_workitems : num_ready;
        source_indices.resize(num_indices);
        for (int source_index{}; source_index < num_indices; /* nothing */) {
          auto any{ false };
          for (int list_index{}; list_index < num_list_indices; ++list_index) {
            const auto& indexState =
              indexStates.list.at(list_start_index + list_index);
            if (indexState.ready_state()) {
              source_indices.at(source_index++) = indexState.sourceIndex;
              list_indices.insert(indexState.sourceIndex.listIndex); // logging
              any = true;
              if (source_index >= num_indices) break;
            }
          }
          if (!any) {
            source_indices.resize(source_index);
            break;
          }
        }
      } else {
        source_indices.resize(0);
      }
      if (source_indices.empty()) {
        std::cerr << "  fill " << stream_index << ": empty " << std::endl;
        return false;
      }
      std::cerr << "  fill " << stream_index << ":"
                << " added " << source_indices.size() << " sources"
                << " from " << list_indices.size() << " sourcelists"
                << " (" << list_start_index << " - "
                << list_start_index + num_list_indices - 1 << ")"
                << std::endl;
      return true;
    }

    void allocCopy(const SourceCompatibilityLists& sources) {
      hipError_t err = hipSuccess;

      auto num_sources = source_indices.size();
      assert(num_sources > 0);
      // alloc source indices
      if (!device_sources) {
        auto sources_bytes = num_sources * sizeof(SourceCompatibilityData);
        err = hipMallocAsync((void **)&device_sources, sources_bytes, stream);
        if (err != hipSuccess) {
          fprintf(stderr, "Failed to allocate stream %d sources, error: %s\n",
            stream_index, hipGetErrorString(err));
          throw std::runtime_error("failed to allocate sources");
        }
      }

      // copy source indices
      int index{};
      for (const auto& sourceIndex: source_indices) {
        err = hipMemcpyAsync(&device_sources[index++],
          &sources.at(sourceIndex.listIndex).at(sourceIndex.index),
          sizeof(SourceCompatibilityData), hipMemcpyHostToDevice, stream);
        if (err != hipSuccess) {
          fprintf(stderr, "Failed to copy stream %d source, error: %s\n",
            stream_index, hipGetErrorString(err));
          throw std::runtime_error("failed to copy source");
        }
      }
      
      // alloc results
      if (!device_results) {
        auto results_bytes = num_sources * sizeof(result_t);
        err = hipMallocAsync((void **)&device_results, results_bytes, stream);
        if (err != hipSuccess) {
          fprintf(stderr, "Failed to allocate stream %d results, error: %s\n",
            stream_index, hipGetErrorString(err));
          throw std::runtime_error("failed to allocate results");
        }
      }
    }

    auto hasWorkRemaining() const {
      return !source_indices.empty();
    }

    SourceCompatibilityData* device_sources = nullptr;
    int num_sources;      // # of entries in device_sources
    int list_start_index; // starting index in SourceCompatibiliityLists
    int num_list_indices; // # of above list entries we are concerned with
    int stream_index;
    bool running = false;
    result_t *device_results = nullptr;
    hipStream_t stream = nullptr;
    std::vector<SourceIndex> source_indices;
  }; // struct KernelData

  //////////

  /*
  bool hasAnyWorkRemaining(const std::vector<KernelData>& kernelData) {
    for (const auto& kd: kernelData) {
      if (kd.hasWorkRemaining()) return true;
    }
    return false;
  }
  */
  bool getRunningComplete(const std::vector<KernelData>& kernelVec,
    int& index)
  {
    // it would be better here to start at index+1 and wrap
    for (auto i{ 0u }; i < kernelVec.size(); ++i) {
      const auto& kd = kernelVec[i];
      if (kd.running && (hipSuccess == hipStreamQuery(kd.stream))) {
        index = i;
        return true;
      }
    }
    return false;
  }

  bool getNextWithWorkRemaining(const std::vector<KernelData>& kernelVec,
    int& current)
  {
    if (!getRunningComplete(kernelVec, current)) {
      bool wrapped = false;
      do {
        if (++current >= kernelVec.size()) {
          current = 0;
          if (wrapped) return false;
          wrapped = true;
        }
      } while (!kernelVec.at(current).hasWorkRemaining());
    }
    return true;
  }

  void runKernel(KernelData& kd) {
    auto num_sources = kd.source_indices.size();
    int threadsPerBlock = 32;
    int blocksPerGrid = (num_sources + threadsPerBlock - 1) / threadsPerBlock;
    fprintf(stderr, "  kernel %d launch with %d blocks of %d threads...\n",
      kd.stream_index, blocksPerGrid, threadsPerBlock);
    
    kd.running = true;
    kernel<<<blocksPerGrid, threadsPerBlock, 0, kd.stream>>>(kd.device_sources,
      num_sources, PCD.device_xorSources, PCD.xorSourceList.size(),
      kd.device_results);
  }

  auto getKernelResults(KernelData& kernel) {
    auto num_sources = kernel.source_indices.size();
    ResultList results(num_sources);
    auto results_bytes = num_sources * sizeof(result_t);
    hipStreamSynchronize(kernel.stream);
    hipError_t err = hipMemcpyAsync(results.data(), kernel.device_results,
      results_bytes, hipMemcpyDeviceToHost, kernel.stream);
    kernel.running = false;
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to copy results from device -> host, error: %s\n",
              hipGetErrorString(err));
      throw std::runtime_error("failed to copy results from device -> host");
    }
    return results;
  }

  /*
  auto increment(int cur, int num) {
    if (++cur >= num) cur = 0;
    return cur;
  }
  */

  void showAllNumReady(const std::vector<KernelData>& kernels,
    const IndexStates& indexStates)
  {
    for (auto& k: kernels) {
      std::cerr << "  kernel " << k.stream_index << ": " 
                << k.num_ready(indexStates) << std::endl;
    }
  }

#if 0
  auto* allocCopySources(const SourceCompatibilityLists& sources) {
    // alloc sources
    hipError_t err = hipSuccess;
    auto sources_bytes = num_sources * sizeof(SourceCompatibilityData);
    SourceCompatibilityData* device_sources;
    err = hipMallocAsync((void **)&device_sources, sources_bytes, stream);
        if (err != hipSuccess) {
          fprintf(stderr, "Failed to allocate stream %d sources, error: %s\n",
            stream_index, hipGetErrorString(err));
          throw std::runtime_error("failed to allocate sources");
        }
      }

    // copy sources
      int index{};
      for (const auto& sourceIndex: source_indices) {
        err = hipMemcpyAsync(&device_sources[index++],
          &sources.at(sourceIndex.listIndex).at(sourceIndex.index),
          sizeof(SourceCompatibilityData), hipMemcpyHostToDevice, stream);
        if (err != hipSuccess) {
          fprintf(stderr, "Failed to copy stream %d source, error: %s\n",
            stream_index, hipGetErrorString(err));
          throw std::runtime_error("failed to copy source");
        }
      }
#endif

} // anonymous namespace

namespace cm {

void filterCandidatesCuda(int sum) {
  using namespace std::chrono;

  std::cerr << "++filterCandidatesCuda" << std::endl;

  const auto& sources = allSumsCandidateData.at(sum - 2).sourceCompatLists;
  //auto device_soruces = allocCopySources(sources);

  //constexpr
  const int num_streams = KernelData::getNumStreams(sources.size());
  std::vector<KernelData> kernels(num_streams);
  KernelData::init(kernels, sources.size());

  IndexStates indexStates{ sources.size() };
  //auto first{ true };
  int total_compatible{};
  int current_kernel = -1;
  while (getNextWithWorkRemaining(kernels, current_kernel)) {
    auto& kd = kernels.at(current_kernel);
    if (!kd.running) {
      if (kd.fillSourceIndices(indexStates)) {
        // TODO: move alloc to separate func outside loop
        // consider copying all source data on stream0, 
        // and only copy indices array here
        kd.allocCopy(sources);
        runKernel(kd);
      }
      continue;
    }

    auto t0 = high_resolution_clock::now();
    auto results = getKernelResults(kd);
    auto t1 = high_resolution_clock::now();
    auto d = duration_cast<milliseconds>(t1 - t0).count();

    /*
    std::cerr << "**BEFORE UPDATE" << std::endl;
    showAllNumReady(kernels, indexStates);
    std::cerr << "-----------" << std::endl;
    */

    auto num_compatible = indexStates.update(kd.source_indices, results, sources);
    total_compatible += num_compatible;

    /*
    std::cerr << "**AFTER UPDATE" << std::endl;
    showAllNumReady(kernels, indexStates);
    std::cerr << "-----------" << std::endl;
    */

    std::cerr << "  kernel " << current_kernel << " done"
#ifdef DEBUG
              << ", done: " << kd.num_done(indexStates)
              << ", compatible reported: " << num_compatible
              << " actual:" << kd.num_compatible(indexStates)
              << ", total compatible: " << total_compatible
              << ", remaining: " << kd.num_ready(indexStates)
#endif
              << " - " << d << "ms" << std::endl;
#ifdef DEBUG
    assert(kd.num_list_indices == kd.num_ready(indexStates) +
      kd.num_compatible(indexStates) + kd.num_done(indexStates));
#endif
  }
  std::cerr << "total compatible: " << total_compatible << " of "
            << sources.size() << std::endl;

  /*
  //  auto num_sources = count(compatLists);
  auto source_bytes = num_sources * sizeof(SourceCompatibilityData);

  auto ac0 = high_resolution_clock::now();
  // begin alloc_copy 

  // end alloc-copy
  auto ac1 = high_resolution_clock::now();
  auto dur_ac = duration_cast<milliseconds>(ac1 - ac0).count();
  std::cerr << "  alloc/copy " << compatLists.size() << " compatLists"
            << " (" << num_sources << ") done - " << dur_ac << "ms"
            << std::endl;
  */

//#define IMMEDIATE_RESULTS
#ifdef IMMEDIATE_RESULTS
  std::vector<result_t> results;
  results.resize(num_source);
  err = hipMemcpy(results.data(), device_results, results_bytes,
                   hipMemcpyDeviceToHost, stream);

  auto& indexComboListMap = allSumsCandidateData.at(sum - 2).indexComboListMap;
  int num_compat_combos{};
  int num_compat_sourcelists{};
  index = 0;
  int list_index{};
  for (const auto& compatList: compatLists) {
    int result_index{ index };
    for (const auto& source: compatList) {
      if (results.at(result_index)) {
        ++num_compat_sourcelists;
        num_compat_combos += indexComboListMap.at(list_index).size();
        break;
      }
      result_index++;
    }
    index += compatList.size();
    ++list_index;
  }
  int num_compat_results = std::accumulate(results.cbegin(), results.cend(), 0,
    [](int num_compatible, result_t result) mutable {
      if (result) num_compatible++;
      return num_compatible;
    });
  std::cerr << "  results: " << results.size()
    << ", compat results: " << num_compat_results
    << ", compat sourcelists: " << num_compat_sourcelists
    << ", compat combos: " << num_compat_combos
    << std::endl;

  err = hipFree(device_results);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device results (error code %s)!\n",
      hipGetErrorString(err));
    throw std::runtime_error("failed to free device results");
  }

  /*
  err = hipFree(device_xorSources);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device xorSources (error code %s)!\n",
      hipGetErrorString(err));
    throw std::runtime_error("failed to free device xorSources");
  }
  */

  err = hipFree(device_compatList);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device compatList (error code %s)!\n",
      hipGetErrorString(err));
    throw std::runtime_error("failed to free device compatList");
  }
#endif // IMMEDIATE_RESULTS

  std::cerr << "--filterCandidatesCuda" << std::endl;
}

[[nodiscard]]
XorSource* cuda_allocCopyXorSources(const XorSourceList& xorSourceList,
  const std::vector<int> sortedIndices)
{
  auto num_xorSources = xorSourceList.size();
  auto xorSources_bytes = num_xorSources * sizeof(XorSource);
  XorSource *device_xorSources = nullptr;
  hipError_t err = hipMalloc((void **)&device_xorSources, xorSources_bytes);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device xorSources, errror: %s\n",
      hipGetErrorString(err));
    throw std::runtime_error("failed to allocate device xorSources");
  }
  for (auto i{ 0u }; i < sortedIndices.size(); ++i) {
    err = hipMemcpyAsync(&device_xorSources[i], &xorSourceList.at(sortedIndices[i]),
      sizeof(XorSource), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to copy xorSources host -> device, error: %s\n",
              hipGetErrorString(err));
      throw std::runtime_error("failed to copy xorSources host -> device");
    }
  }
  return device_xorSources;
}

} // namespace cm
