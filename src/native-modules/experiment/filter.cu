#include "hip/hip_runtime.h"
// filter.cu

#include <algorithm>
#include <chrono>
#include <exception>
#include <future>
#include <iostream>
#include <limits>
#include <numeric>
#include <optional>
#include <span>
#include <thread>
#include <tuple>
#include <utility> // pair
#include <hip/hip_runtime.h>
#include "filter.cuh"
#include "stream-data.h"
#include "merge-filter-data.h"
#include "util.h"

//#define LOGGING

namespace cm {

namespace {

/*
__device__ bool variation_indices_shown = false;

__device__ void print_variation_indices(
    const device::VariationIndices* __restrict__ variation_indices) {
  if (variation_indices_shown)
    return;
  printf("device:\n");
  for (int s{}; s < kNumSentences; ++s) {
    const auto& vi = variation_indices[s];
    for (index_t v{}; v < vi.num_variations; ++v) {
      const auto n = vi.num_combo_indices[v];
      printf("sentence %d, variation %d, indices: %d\n", s, v, n);
    }
  }
  variation_indices_shown = true;
}
*/

template <typename T> __device__ __forceinline__ T load(const T* addr) {
  return *(const volatile T*)addr;
}

template <typename T> __device__ __forceinline__ void store(T* addr, T val) {
  *(volatile T*)addr = val;
}

// Test if the supplied source contains both of the primary sources described
// by any of the supplied source descriptor pairs.
__device__ bool source_contains_any_descriptor_pair(
    const SourceCompatibilityData& source,
    const UsedSources::SourceDescriptorPair* __restrict__ src_desc_pairs,
    const unsigned num_src_desc_pairs) {

  __shared__ bool contains_both;
  if (!threadIdx.x) contains_both = false;
  // one thread per src_desc_pair
  for (unsigned idx{}; idx * blockDim.x < num_src_desc_pairs; ++idx) {
    __syncthreads();
    if (contains_both) return true;
    const auto pair_idx = idx * blockDim.x + threadIdx.x;
    if (pair_idx < num_src_desc_pairs) {
      if (source.usedSources.has(src_desc_pairs[pair_idx])) {
        contains_both = true;
      }
    }
  }
  return false;
}

__global__ void get_compatible_sources_kernel(
    const SourceCompatibilityData* __restrict__ sources,
    const unsigned num_sources,
    const UsedSources::
        SourceDescriptorPair* __restrict__ incompatible_src_desc_pairs,
    const unsigned num_src_desc_pairs,
    compat_src_result_t* __restrict__ results) {
  // one block per source
  for (unsigned idx{blockIdx.x}; idx < num_sources; idx += gridDim.x) {
    const auto& source = sources[idx];
    // I don't understand why this is required, but fails synccheck and
    // produces wobbly results without.
    __syncthreads();
    if (!source_contains_any_descriptor_pair(
          source, incompatible_src_desc_pairs, num_src_desc_pairs)) {
      if (!threadIdx.x) {
        results[idx] = 1;
      }
    }
  }
}

extern __shared__ unsigned dynamic_shared[];

/*
__device__ bool is_source_OR_compatible_old(const SourceCompatibilityData& source,
    const unsigned num_or_args,
    const device::OrSourceData* __restrict__ or_arg_sources,
    const unsigned num_or_arg_sources) {
  //  extern __shared__ result_t or_arg_results[];
  // ASSUMPTION: # of --or args will always be smaller than block size.
  if (threadIdx.x < num_or_args) {
    or_arg_results[threadIdx.x] = 0;
  }
  const auto chunk_size = blockDim.x;
  const auto chunk_max = num_or_arg_sources;
  for (unsigned chunk_idx{}; chunk_idx * chunk_size < chunk_max; ++chunk_idx) {
    __syncthreads();
    const auto or_arg_src_idx = chunk_idx * chunk_size + threadIdx.x;
    if (or_arg_src_idx < num_or_arg_sources) {
      const auto& or_src = or_arg_sources[or_arg_src_idx];
      // NB! order of a,b in a.isOrCompat(b) here matters!
      if (or_src.src.isOrCompatibleWith(source)) {
        or_arg_results[or_src.or_arg_idx] = 1;
      }
    }
  }
  __syncthreads();
  if (!threadIdx.x) {
    bool compat_with_all{true};
    for (int i{}; i < num_or_args; ++i) {
      if (!or_arg_results[i]) {
        compat_with_all = false;
        break;
      }
    }
    return compat_with_all;
  }
  return false;
}

  if (num_or_args > 0) {
    // source must also be OR compatible with at least one source
    // of each or_arg
    if (is_source_OR_compatible(
            source, num_or_args, or_arg_sources, num_or_arg_sources)) {
      is_or_compat = true;
    }
    __syncthreads();
    if (!is_or_compat) {
      continue;
    }
  }
*/

struct SmallestSpansResult {
  bool skip;
  ComboIndexSpanPair spans;
};
using smallest_spans_result_t = SmallestSpansResult;

// variation_indices is an optimization. it allows us to restrict comparisons
// of a candidate source to only those xor_sources that have the same (or no)
// variation for each sentence - since a variation mismatch will alway result
// in comparison failure.
//
// individual xor_src indices are potentially (and often) duplicated in the
// variation_indices lists. for example, if a particular compound xor_src has
// variations S1:V1 and S2:V3, its xor_src_idx will appear in the variation
// indices lists for both of sentences 1 and 2.
//
// because of this, we only need to compare a candidate source with the
// xor_sources that have the same (or no) variations for a *single* sentence.
//
// the question then becomes, which sentence should we choose?
//
// the answer: the one with the fewest indices! (which results in the fewest
// comparisons). that's what this function determines.
//
__device__ smallest_spans_result_t get_smallest_src_index_spans(
    const SourceCompatibilityData& source,
    const device::VariationIndices* __restrict__ variation_indices) {
  index_t fewest_indices{std::numeric_limits<index_t>::max()};
  int sentence_with_fewest{-1};
  for (int s{}; s < kNumSentences; ++s) {
    // if there are no xor_sources that contain a primary source from this
    // sentence, skip it. (it would result in num_indices == all_indices which
    // is the worst case).
    const auto& vi = variation_indices[s];
    if (!vi.num_variations) continue;

    // if the candidate source has no primary source from this sentence, skip
    // it. (same reason as above).
    const auto variation = source.usedSources.variations[s] + 1;
    if (!variation) continue;

    // sum the xor_src indices that have no variation (index 0), with those
    // that have the same variation as the candidate source, for this sentence.
    // remember the sentence with the smallest sum.
    const auto num_indices =
        vi.num_combo_indices[0] + vi.num_combo_indices[variation];
    if (num_indices < fewest_indices) {
      fewest_indices = num_indices;
      sentence_with_fewest = s;
      if (!fewest_indices) break;
    }
  }
  if (sentence_with_fewest < 0) {
    // there are no sentences from which both the candidate source and any
    // xor_source contain a primary source. we can skip xor-compat checks
    // since they will all succeed.
    return {true};
  }
  if (!fewest_indices) {
    // both the candidate source and all xor_sources contain a primary source
    // from sentence_with_fewest, but all xor_sources use a different variation
    // than the candidate source. we can skip all xor-compat checks since they
    // will all fail due to variation mismatch.
    return {true};
  }
  const auto variation =
      source.usedSources.variations[sentence_with_fewest] + 1;
  const auto& vi = variation_indices[sentence_with_fewest];
  return {false,  //
      std::make_pair(vi.get_index_span(0), vi.get_index_span(variation))};
}

__device__ uint64_t get_flat_idx(uint64_t block_idx) {
  return block_idx * uint64_t(blockDim.x) + threadIdx.x;
}

__device__ combo_index_t get_xor_combo_index(
    uint64_t flat_idx, const ComboIndexSpanPair& idx_spans) {
  if (flat_idx < idx_spans.first.size()) { return idx_spans.first[flat_idx]; }
  flat_idx -= idx_spans.first.size();
  assert(flat_idx < idx_spans.second.size());
  return idx_spans.second[flat_idx];
}

__device__ auto get_combo_index(uint64_t flat_idx,
    const index_t* __restrict__ idx_list_sizes, unsigned num_idx_lists) {
  combo_index_t combo_idx{};
  for (int list_idx{int(num_idx_lists) - 1}; list_idx >= 0; --list_idx) {
    const auto idx_list_size = idx_list_sizes[list_idx];
    if (combo_idx) combo_idx *= idx_list_size;
    combo_idx += flat_idx % idx_list_size;
    flat_idx /= idx_list_size;
  }
  return combo_idx;
}

const int kXorChunkIdx = 0;
const int kOrChunkIdx = 1;
const int kXorResultsIdx = 2;
const int kNumSharedIndices = 4;  // 32-bit align

// TODO: templatize this & is_source_XOR_compatible
// Test if a source is OR compatible with the OR source specified by the
// supplied combo index and index lists.
__device__ bool is_source_OR_compatible(const SourceCompatibilityData& source,
    combo_index_t or_combo_idx,
    const SourceCompatibilityData* __restrict__ or_src_lists,
    const index_t* __restrict__ or_src_list_start_indices,
    const index_t* __restrict__ or_idx_lists,
    const index_t* __restrict__ or_idx_list_start_indices,
    const index_t* __restrict__ or_idx_list_sizes, unsigned num_or_idx_lists) {
  for (unsigned list_idx{}; list_idx < num_or_idx_lists; ++list_idx) {
    const auto or_src_list =
        &or_src_lists[or_src_list_start_indices[list_idx]];
    const auto idx_list = &or_idx_lists[or_idx_list_start_indices[list_idx]];
    const auto idx_list_size = or_idx_list_sizes[list_idx];
    const auto or_src_idx = idx_list[or_combo_idx % idx_list_size];
    const auto& or_src = or_src_list[or_src_idx];
    if (!or_src.isOrCompatibleWith(source)) return false;
    or_combo_idx /= idx_list_size;
  }
  return true;
}

// Get the next block-sized chunk of OR sources and test them for variation-
// compatibililty with the supplied XOR result index for the specified XOR
// chunk. And for OR-compatibility with the supplied (??) source?
// Return true if at least one OR source is compatible.
__device__ bool get_next_OR_sources_chunk(const SourceCompatibilityData& source,
    unsigned or_chunk_idx, const combo_index_t xor_combo_idx,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes, unsigned num_idx_lists,
    const SourceCompatibilityData* __restrict__ or_src_lists,
    const index_t* __restrict__ or_src_list_start_indices,
    const index_t* __restrict__ or_idx_lists,
    const index_t* __restrict__ or_idx_list_start_indices,
    const index_t* __restrict__ or_idx_list_sizes,
    const unsigned num_or_idx_lists, const unsigned num_or_sources,
    const combo_index_t* __restrict__ or_variations,
    const unsigned num_or_variations, result_t* __restrict__ or_results) {
  const auto block_size = blockDim.x;
  __shared__ bool is_variation_compat;
  __shared__ bool is_or_compat;
  if (!threadIdx.x) {
    is_or_compat = false;
    is_variation_compat = false;
    or_results[threadIdx.x] = 0;
  }
  __syncthreads();
  const auto flat_idx = get_flat_idx(or_chunk_idx);
  const auto or_combo_idx =
      get_combo_index(flat_idx, or_idx_list_sizes, num_or_idx_lists);
  if (flat_idx < num_or_sources) {
#if 0
    if (are_variations_compatible(
            or_combo_idx, or_src_lists, xor_combo_idx, xor_src_lists)) {
#endif
      is_variation_compat = true;
//    }
  }
  __syncthreads();
  if (is_variation_compat) {
    if (is_source_OR_compatible(source, or_combo_idx, or_src_lists,
            or_src_list_start_indices, or_idx_lists, or_idx_list_start_indices,
            or_idx_list_sizes, num_or_idx_lists)) {
      is_or_compat = true;
      or_results[threadIdx.x] = 1;
    }
  }
  __syncthreads();
  return is_variation_compat && is_or_compat;
}

// for one XOR result
__device__ bool get_next_compatible_OR_sources(
    const SourceCompatibilityData& source, unsigned or_chunk_idx,
    combo_index_t xor_combo_idx,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes, unsigned num_xor_idx_lists,
    const SourceCompatibilityData* __restrict__ or_src_lists,
    const index_t* __restrict__ or_src_list_start_indices,
    const index_t* __restrict__ or_idx_lists,
    const index_t* __restrict__ or_idx_list_start_indices,
    const index_t* __restrict__ or_idx_list_sizes,
    const unsigned num_or_idx_lists, const unsigned num_or_sources,
    const combo_index_t* __restrict__ or_variations,
    const unsigned num_or_variations, result_t* __restrict__ or_results) {
  const auto block_size = blockDim.x;
  __shared__ bool any_or_compat;
  unsigned* or_chunk_idx_ptr = &dynamic_shared[kOrChunkIdx];
  if (!threadIdx.x) any_or_compat = false;
  __syncthreads();
  for (; or_chunk_idx * block_size < num_or_sources; ++or_chunk_idx) {
    if (get_next_OR_sources_chunk(source, or_chunk_idx, xor_combo_idx,
            xor_src_lists, xor_src_list_start_indices, xor_idx_lists,
            xor_idx_list_start_indices, xor_idx_list_sizes, num_xor_idx_lists,
            or_src_lists, or_src_list_start_indices, or_idx_lists,
            or_idx_list_start_indices, or_idx_list_sizes, num_or_idx_lists,
            num_or_sources, or_variations, num_or_variations, or_results)) {
      any_or_compat = true;
    }
    __syncthreads();
    if (any_or_compat) break;
  }
  if (!threadIdx.x) { *or_chunk_idx_ptr = or_chunk_idx; }
  return any_or_compat;
}

__device__ auto next_xor_result_idx(
    unsigned result_idx, const result_t* __restrict__ xor_results) {
  const auto block_size = blockDim.x;
  while ((result_idx < block_size) && !xor_results[result_idx])
    result_idx++;
  return result_idx;
}

// for all XOR results
__device__ bool get_next_compatible_OR_sources(
    const SourceCompatibilityData& source, unsigned or_chunk_idx,
    unsigned xor_chunk_idx, const result_t* __restrict__ xor_results,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes, unsigned num_xor_idx_lists,
    const ComboIndexSpanPair& idx_spans,
    const SourceCompatibilityData* __restrict__ or_src_lists,
    const index_t* __restrict__ or_src_list_start_indices,
    const index_t* __restrict__ or_idx_lists,
    const index_t* __restrict__ or_idx_list_start_indices,
    const index_t* __restrict__ or_idx_list_sizes,
    const unsigned num_or_idx_lists, const unsigned num_or_sources,
    const combo_index_t* __restrict__ or_variations,
    const unsigned num_or_variations, result_t* __restrict__ or_results) {
  const auto block_size = blockDim.x;
  __shared__ bool any_or_compat;
  unsigned* xor_results_idx = &dynamic_shared[kXorResultsIdx];
  unsigned* or_chunk_idx_ptr = &dynamic_shared[kOrChunkIdx];
  const auto num_xor_indices = idx_spans.first.size() + idx_spans.second.size();
  if (!threadIdx.x) {
    if (!or_chunk_idx) {
      *xor_results_idx = next_xor_result_idx(0, xor_results);
    }
    any_or_compat = false;
  }
  __syncthreads();
  while (*xor_results_idx < block_size) {
    const auto xor_flat_idx = get_flat_idx(xor_chunk_idx);
    assert(xor_flat_idx < num_xor_indices);
    const auto xor_combo_idx = get_xor_combo_index(xor_flat_idx, idx_spans);
    if (get_next_compatible_OR_sources(source, or_chunk_idx, xor_combo_idx,
            xor_src_lists, xor_src_list_start_indices, xor_idx_lists,
            xor_idx_list_start_indices, xor_idx_list_sizes, num_xor_idx_lists,
            or_src_lists, or_src_list_start_indices, or_idx_lists,
            or_idx_list_start_indices, or_idx_list_sizes, num_or_idx_lists,
            num_or_sources, or_variations, num_or_variations, or_results)) {
      any_or_compat = true;
    }
    __syncthreads();
    if (any_or_compat) break; // sync failure here?
    if (!threadIdx.x) {
      *xor_results_idx = next_xor_result_idx(*xor_results_idx + 1, xor_results);
    }
    __syncthreads();
  }
  // NOT SURE ABOUT THIS but seems maybe right. possibly. hopefully?
  if (!threadIdx.x) *or_chunk_idx_ptr = 0;
  return any_or_compat;
}

// Test if a source is XOR compatible with the XOR source specified by the
// supplied combo index and index lists.
__device__ bool is_source_XOR_compatible(const SourceCompatibilityData& source,
    combo_index_t combo_idx,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes,
    unsigned num_xor_idx_lists) {

  for (unsigned list_idx{}; list_idx < num_xor_idx_lists; ++list_idx) {
    const auto xor_src_list =
        &xor_src_lists[xor_src_list_start_indices[list_idx]];
    const auto idx_list = &xor_idx_lists[xor_idx_list_start_indices[list_idx]];
    const auto idx_list_size = xor_idx_list_sizes[list_idx];
    const auto xor_src_idx = idx_list[combo_idx % idx_list_size];
    if (!source.isXorCompatibleWith(xor_src_list[xor_src_idx])) return false;
    combo_idx /= idx_list_size;
  }
  return true;
}

// Get the next block-sized chunk of XOR sources and test them for 
// compatibility with the supplied source.
// Return true if at least one XOR source is compatible.
__device__ bool get_next_XOR_sources_chunk(
    const SourceCompatibilityData& source, const unsigned xor_chunk_idx,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes,
    const unsigned num_xor_idx_lists, const ComboIndexSpanPair& idx_spans,
    result_t* __restrict__ xor_results) {
  __shared__ bool is_xor_compat;
  const auto block_size = blockDim.x;
  const auto num_xor_indices = idx_spans.first.size() + idx_spans.second.size();
  if (!threadIdx.x) {
    is_xor_compat = false;
    xor_results[threadIdx.x] = 0;
  }
  __syncthreads();
  const auto xor_flat_idx = get_flat_idx(xor_chunk_idx);
  if (xor_flat_idx < num_xor_indices) {
    const auto xor_combo_idx = get_xor_combo_index(xor_flat_idx, idx_spans);
    if (is_source_XOR_compatible(source, xor_combo_idx, xor_src_lists,
            xor_src_list_start_indices, xor_idx_lists,
            xor_idx_list_start_indices, xor_idx_list_sizes,
            num_xor_idx_lists)) {
      is_xor_compat = true;
      xor_results[threadIdx.x] = 1;
    }
  }
  __syncthreads();
  return is_xor_compat;
}

// Loop through block-sized chunks of XOR sources until we find one that 
// contains at leasst one XOR source that is compatibile with the supplied
// source, or until all XOR sources are exhausted.
// Return true if at least one XOR source is compatible.
__device__ bool get_next_compatible_XOR_sources(
    const SourceCompatibilityData& source, unsigned xor_chunk_idx,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes,
    const unsigned num_xor_idx_lists, const ComboIndexSpanPair& idx_spans,
    result_t* __restrict__ xor_results) {
  unsigned* xor_chunk_idx_ptr = &dynamic_shared[kXorChunkIdx];
  __shared__ bool any_xor_compat;
  const auto num_indices = idx_spans.first.size() + idx_spans.second.size();
  const auto block_size = blockDim.x;
  if (!threadIdx.x) any_xor_compat = false;
  __syncthreads();
  for (; xor_chunk_idx * block_size < num_indices; ++xor_chunk_idx) {
    if (get_next_XOR_sources_chunk(source, xor_chunk_idx, xor_src_lists,
            xor_src_list_start_indices, xor_idx_lists,
            xor_idx_list_start_indices, xor_idx_list_sizes, num_xor_idx_lists,
            idx_spans, xor_results)) {
      any_xor_compat = true;
    }
    __syncthreads();
    if (any_xor_compat) break;
  }
  if (!threadIdx.x) { *xor_chunk_idx_ptr = xor_chunk_idx; }
  return any_xor_compat;
}

// Test if the supplied source is:
// * XOR-compatible with any of the supplied XOR sources
// * OR-compatible with any of the supplied OR sources which are
//   variation-compatible with the compatible XOR source.
//
// In other words:
// For each XOR source that is XOR-compatible with Source
//   For each OR source that is variation-compatible with XOR source
//     If OR source is OR-compatible with Source
//       is_compat = true;
__device__ bool is_compat_loop(const SourceCompatibilityData& source,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes,
    const unsigned num_xor_idx_lists, const ComboIndexSpanPair& idx_spans,
    const SourceCompatibilityData* __restrict__ or_src_lists,
    const index_t* __restrict__ or_src_list_start_indices,
    const index_t* __restrict__ or_idx_lists,
    const index_t* __restrict__ or_idx_list_start_indices,
    const index_t* __restrict__ or_idx_list_sizes,
    const unsigned num_or_idx_lists, const unsigned num_or_sources,
    const combo_index_t* __restrict__ or_variations,
    const unsigned num_or_variations) {
  const auto block_size = blockDim.x;
  __shared__ bool any_xor_compat;
  __shared__ bool any_or_compat;
  unsigned* xor_chunk_idx_ptr = &dynamic_shared[kXorChunkIdx];
  unsigned* or_chunk_idx_ptr = &dynamic_shared[kOrChunkIdx];
  result_t* xor_results = (result_t*)&dynamic_shared[kNumSharedIndices];
  result_t* or_results = (result_t*)&xor_results[block_size];
  const unsigned num_xor_indices =
      idx_spans.first.size() + idx_spans.second.size();
  if (!threadIdx.x) {
    *xor_chunk_idx_ptr = 0;
    *or_chunk_idx_ptr = 0;
    // TODO: might do better with an enum here, fewer back-to-back __syncthreads
    // calls potentially
    any_xor_compat = false;
    any_or_compat = !num_or_sources;
  }
  //unsigned* chunk_idx_ptr = xor_chunk_idx_ptr;
  //unsigned chunk_max = num_xor_indices;
  __syncthreads();
  for (;;) {
    /*
    if (!any_xor_compat) {
      if (*xor_chunk_idx_ptr * block_size >= num_xor_indices) break;
      //chunk_idx_ptr = xor_chunk_idx_ptr;
      //chunk_max = num_xor_indices;
    } else {
      assert(!any_or_compat);
      if (*or_chunk_idx_ptr * block_size >= num_or_sources) break;
      //chunk_idx_ptr = or_chunk_idx;
      //chunk_max = num_or_sources;
    }
    // if (*chunk_idx_ptr * block_size >= chunk_max) break;
    */
    if (!any_xor_compat) {
      if (*xor_chunk_idx_ptr * block_size >= num_xor_indices) break;
      if (get_next_compatible_XOR_sources(source, *xor_chunk_idx_ptr,
              xor_src_lists, xor_src_list_start_indices, xor_idx_lists,
              xor_idx_list_start_indices, xor_idx_list_sizes, num_xor_idx_lists,
              idx_spans, xor_results)) {
        any_xor_compat = true;
      }
    }
    __syncthreads();
    //if (any_xor_compat) return true;
#if 1
    if (any_xor_compat && !any_or_compat) {
      if (*or_chunk_idx_ptr * block_size >= num_or_sources) break;
      if (get_next_compatible_OR_sources(source, *or_chunk_idx_ptr,
              *xor_chunk_idx_ptr, xor_results, xor_src_lists,
              xor_src_list_start_indices, xor_idx_lists,
              xor_idx_list_start_indices, xor_idx_list_sizes, num_xor_idx_lists,
              idx_spans, or_src_lists, or_src_list_start_indices, or_idx_lists,
              or_idx_list_start_indices, or_idx_list_sizes, num_or_idx_lists,
              num_or_sources, or_variations, num_or_variations, or_results)) {
        any_or_compat = true;
      }
    }
    __syncthreads();
    if (any_or_compat) return true;
#endif
  }
  return false;
}

// explain better:
// Find sources that are:
// * XOR compatible with any of the supplied XOR sources, and
// * OR compatible with any of the supplied OR sources, which must in turn be
// * variation-compatible with the XOR source.
//
// Find compatible XOR source -> compare with variation-compatible OR sources.
__global__ void filter_kernel(
    const SourceCompatibilityData* __restrict__ src_list,
    const unsigned num_sources,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes,
    const unsigned num_xor_idx_lists,
    const device::VariationIndices* __restrict__ xor_variation_indices,
    const SourceIndex* __restrict__ src_indices,
    const index_t* __restrict__ src_list_start_indices,
    const compat_src_result_t* __restrict__ compat_src_results,
    const SourceCompatibilityData* __restrict__ or_src_lists,
    const index_t* __restrict__ or_src_list_start_indices,
    const index_t* __restrict__ or_idx_lists,
    const index_t* __restrict__ or_idx_list_start_indices,
    const index_t* __restrict__ or_idx_list_sizes,
    const unsigned num_or_idx_lists, const unsigned num_or_sources,
    const combo_index_t* __restrict__ or_variations,
    const unsigned num_or_variations, result_t* __restrict__ results,
    int stream_idx) {
  // TODO __global__ compat int, cuda_memset to zero before launching kernel
  const auto block_size = blockDim.x;
  result_t* compat_results = (result_t*)&dynamic_shared[kNumSharedIndices];
  __shared__ bool is_compat;
  if (!threadIdx.x) {
    is_compat = false;
  }
  // for each source (one block per source)
  for (unsigned idx{blockIdx.x}; idx < num_sources; idx += gridDim.x) {
    __syncthreads();
    const auto src_idx = src_indices[idx];
    const auto flat_idx =
        src_list_start_indices[src_idx.listIndex] + src_idx.index;
    if (compat_src_results && !compat_src_results[flat_idx]) continue;
    const auto& source = src_list[flat_idx];
    auto smallest = get_smallest_src_index_spans(source, xor_variation_indices);
    if (smallest.skip) continue;

    if (is_compat_loop(source, xor_src_lists, xor_src_list_start_indices,
            xor_idx_lists, xor_idx_list_start_indices, xor_idx_list_sizes,
            num_xor_idx_lists, smallest.spans, or_src_lists,
            or_src_list_start_indices, or_idx_lists, or_idx_list_start_indices,
            or_idx_list_sizes, num_or_idx_lists, num_or_sources, or_variations,
            num_or_variations)) {
      is_compat = true;
    }
    __syncthreads();
    if (is_compat && !threadIdx.x) {
      results[src_idx.listIndex] = 1;
      is_compat = false;
    }
  }
}

/*
// Test if a source is XOR compatible with any of the supplied XOR sources.
__device__ bool is_source_XOR_compatible_with_any(
    const SourceCompatibilityData& source,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes,
    const unsigned num_idx_lists, const ComboIndexSpanPair& idx_spans) {
  __shared__ bool is_xor_compat;
  if (!threadIdx.x) is_xor_compat = false;
  const auto num_indices = idx_spans.first.size() + idx_spans.second.size();
  const auto chunk_size = blockDim.x;
  const auto chunk_max = num_indices;
  __syncthreads();
  // one thread per xor_source
  for (unsigned chunk_idx{}; chunk_idx * chunk_size < chunk_max; ++chunk_idx) {
    // "flat" index, i.e. not yet indexed into appropriate idx_spans array
    const auto flat_idx = chunk_idx * chunk_size + threadIdx.x;
    if (flat_idx < num_indices) {
      const auto combo_idx = get_xor_combo_index(flat_idx, idx_spans);
      if (is_source_XOR_compatible(source, combo_idx, xor_src_lists,
              xor_src_list_start_indices, xor_idx_lists,
              xor_idx_list_start_indices, xor_idx_list_sizes, num_idx_lists)) {
        is_xor_compat = true;
      }
    }
    __syncthreads();
    if (is_xor_compat) return true;
  }
  return false;
}

__global__ void xor_kernel_new(
    const SourceCompatibilityData* __restrict__ src_list,
    const unsigned num_sources,
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes,
    const unsigned num_idx_lists,
    const device::VariationIndices* __restrict__ variation_indices,
    const SourceIndex* __restrict__ src_indices,
    const index_t* __restrict__ src_list_start_indices,
    const compat_src_result_t* __restrict__ compat_src_results,
    result_t* __restrict__ results, int stream_idx) {
  __shared__ bool is_compat;
  if (!threadIdx.x) is_compat = false;
  // for each source (one block per source)
  for (unsigned idx{blockIdx.x}; idx < num_sources; idx += gridDim.x) {
    __syncthreads();
    const auto src_idx = src_indices[idx];
    const auto flat_idx =
        src_list_start_indices[src_idx.listIndex] + src_idx.index;
    if (compat_src_results && !compat_src_results[flat_idx]) continue;
    const auto& source = src_list[flat_idx];
    auto result = get_smallest_src_index_spans(source, variation_indices);
    if (result.skip) continue;
    if (is_source_XOR_compatible_with_any(source, xor_src_lists,
            xor_src_list_start_indices, xor_idx_lists,
            xor_idx_list_start_indices, xor_idx_list_sizes, num_idx_lists,
            result.spans)) {
      is_compat = true;
    }
    __syncthreads();
    if (is_compat && !threadIdx.x) {
      results[src_idx.listIndex] = 1;
      is_compat = false;
    }
  }
}
*/

}  // anonymous namespace

void run_filter_kernel(int threads_per_block, StreamData& stream,
    const MergeFilterData& mfd, const SourceCompatibilityData* device_src_list,
    const compat_src_result_t* device_compat_src_results,
    result_t* device_results, const index_t* device_list_start_indices) {
  int num_sm;
  hipDeviceGetAttribute(&num_sm, hipDeviceAttributeMultiprocessorCount, 0);
  int threads_per_sm;;
  hipDeviceGetAttribute(&threads_per_sm, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);

  stream.is_running = true;
  stream.sequence_num = StreamData::next_sequence_num();

  auto block_size = threads_per_block ? threads_per_block : 768;
  auto blocks_per_sm = threads_per_sm / block_size;
  assert(blocks_per_sm * block_size == threads_per_sm);
  auto grid_size = num_sm * blocks_per_sm;  // aka blocks per grid
  // xor_chunk_idx, or_chunk_idx, xor_result_idx, xor_results, or_results
  // results could probably be moved to global
  auto shared_bytes = 
      kNumSharedIndices * sizeof(unsigned) + block_size * 2 * sizeof(result_t);
  // ensure any async alloc/copies are complete on main thread stream
  hipError_t err = hipStreamSynchronize(hipStreamPerThread);
  assert_cuda_success(err, "run_filter_kernel sync");
  dim3 grid_dim(grid_size);
  dim3 block_dim(block_size);
  stream.xor_kernel_start.record();
  /*
  xor_kernel_new<<<grid_dim, block_dim, shared_bytes, stream.hip_stream>>>(
      device_src_list, stream.src_indices.size(), mfd.device_xor.src_lists,
      mfd.device_xor.src_list_start_indices, mfd.device_xor.idx_lists,
      mfd.device_xor.idx_list_start_indices, mfd.device_xor.idx_list_sizes,
      mfd.host_xor.compat_idx_lists.size(), mfd.device_xor.variation_indices,
      stream.device_src_indices, device_list_start_indices,
      device_compat_src_results, device_results, stream.stream_idx);
  */
  filter_kernel<<<grid_dim, block_dim, shared_bytes, stream.hip_stream>>>(
      // Sources
      device_src_list, stream.src_indices.size(),
      // XOR sources
      mfd.device_xor.src_lists, mfd.device_xor.src_list_start_indices,
      mfd.device_xor.idx_lists, mfd.device_xor.idx_list_start_indices,
      mfd.device_xor.idx_list_sizes, mfd.host_xor.compat_idx_lists.size(),
      mfd.device_xor.variation_indices,
      // Sources again
      stream.device_src_indices, device_list_start_indices,
      // XOR sources again i think, for count > 2
      device_compat_src_results,
      // OR sources
      mfd.device_or.src_lists, mfd.device_or.src_list_start_indices,
      mfd.device_or.idx_lists, mfd.device_or.idx_list_start_indices,
      mfd.device_or.idx_list_sizes, mfd.host_or.compat_idx_lists.size(),
      util::sum_sizes(mfd.host_or.compat_idx_lists),
      mfd.device_or.combo_indices, mfd.host_or.combo_indices.size(),
      device_results, stream.stream_idx);
  stream.xor_kernel_stop.record();
  if constexpr (0) {
    std::cerr << "stream " << stream.stream_idx << " XOR kernel started with " << grid_size
              << " blocks of " << block_size << " threads" << std::endl;
  }
}

void run_get_compatible_sources_kernel(
    const SourceCompatibilityData* device_src_list, unsigned num_sources,
    const UsedSources::SourceDescriptorPair* device_src_desc_pairs,
    unsigned num_src_desc_pairs, compat_src_result_t* device_results) {
  int num_sm;
  hipDeviceGetAttribute(&num_sm, hipDeviceAttributeMultiprocessorCount, 0);
  int threads_per_sm;
  hipDeviceGetAttribute(
      &threads_per_sm, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
  auto block_size = 768;  // aka threads per block
  auto blocks_per_sm = threads_per_sm / block_size;
  assert(blocks_per_sm * block_size == threads_per_sm);
  auto grid_size = num_sm * blocks_per_sm;  // aka blocks per grid
  auto shared_bytes = 0;

  dim3 grid_dim(grid_size);
  dim3 block_dim(block_size);
  // async copies are on thread stream therefore auto synchronized
  hipStream_t stream = hipStreamPerThread;
  get_compatible_sources_kernel<<<grid_dim, block_dim, shared_bytes, stream>>>(
      device_src_list, num_sources, device_src_desc_pairs, num_src_desc_pairs,
      device_results);
}

/*
__device__ bool is_source_OR_compatible(const SourceCompatibilityData& source,
    const unsigned num_or_args,
    const device::OrSourceData* __restrict__ or_arg_sources,
    const unsigned num_or_arg_sources) {
   extern __shared__ result_t or_arg_results[];
  // ASSUMPTION: # of --or args will always be smaller than block size.
  if (threadIdx.x < num_or_args) {
    or_arg_results[threadIdx.x] = 0;
  }
  const auto chunk_size = blockDim.x;
  const auto chunk_max = num_or_arg_sources;
  for (unsigned chunk_idx{}; chunk_idx * chunk_size < chunk_max; ++chunk_idx) {
    __syncthreads();
    const auto or_arg_src_idx = chunk_idx * chunk_size + threadIdx.x;
    if (or_arg_src_idx < num_or_arg_sources) {
      const auto& or_src = or_arg_sources[or_arg_src_idx];
      // NB! order of a,b in a.isOrCompat(b) here matters!
      if (or_src.src.isOrCompatibleWith(source)) {
        or_arg_results[or_src.or_arg_idx] = 1;
      }
    }
  }
  __syncthreads();
  if (!threadIdx.x) {
    bool compat_with_all{true};
    for (int i{}; i < num_or_args; ++i) {
      if (!or_arg_results[i]) {
        compat_with_all = false;
        break;
      }
    }
    return compat_with_all;
  }
  return false;
}

void show_or_arg_counts([[maybe_unused]] unsigned num_or_args) {
  hipError_t err{hipSuccess};

#if 0
  unsigned wtfbbq;
  err = hipMemcpyFromSymbol(
      &wtfbbq, HIP_SYMBOL(device_wtfbbq), sizeof(unsigned));
  assert_cuda_success(err, "cudaMemCopyFromSymbol wtfbbq");
  std::cerr << "wtfbbq: " << wtfbbq << std::endl;
#endif

#ifdef OR_ARG_COUNTS
  unsigned num_compat;
  err = hipMemcpyFromSymbol(
      &num_compat, HIP_SYMBOL(device_num_compat_or_args), sizeof(unsigned));
  assert_cuda_success(err, "cudaMemCopyFromSymbol num_compat");
  std::cerr << "compatible sources: " << num_compat << std::endl;

  unsigned results[MAX_OR_ARGS] = {0};
  auto max_or_args = std::min(MAX_OR_ARGS, (int)num_or_args);
  err = hipMemcpyFromSymbol(results, HIP_SYMBOL(device_incompatible_or_arg_counts),
      max_or_args * sizeof(unsigned));
  assert_cuda_success(err, "cudaMemCopyFromSymbol
incompatible_or_arg_counts"); std::cerr << "incompatible or_args:\n"; for (int
i{}; i < max_or_args; ++i) { std::cerr << " arg" << i << ": " << results[i] <<
std::endl;
  }
#endif
}
*/

/*
// Test if a source is XOR compatible with ANY of the provided xor sources.
__device__ bool is_source_XOR_compatible_with_any(
    const SourceCompatibilityData& source, const ComboIndexSpanPair&
idx_spans, const SourceCompatibilityData* __restrict__ xor_src_lists, const
index_t* __restrict__ xor_src_list_start_indices, const index_t* __restrict__
xor_idx_lists, const index_t* __restrict__ xor_idx_list_start_indices, const
index_t* __restrict__ xor_idx_list_sizes, const unsigned num_idx_lists) {

  __shared__ bool is_xor_compat;
  if (!threadIdx.x) {
    store(&is_xor_compat, false);
  }
  // NOTE: chunk-indexing as used here is necessary for syncthreads() to work
  //   at least on SM_6 hardware (GTX1060), where *all threads* in the block
  //   must execute the synchthreads() call. In later architectures, those
  //   restrictions may be relaxed, but possibly only for "completely exited
  //   (the kernel)" threads, which wouldn't be relevant here anyway (because
  //   we're in a function called from within a loop in a parent kernel).
  //
  //   Therefore, the following is not an equivalent replacement:
  //
  //   for (unsigned flat_idx{threadIdx.x}; flat_idx < num_xor_sources;
  //      flat_idx += blockDim.x) {
  //
  // TODO: not sure all the syncthreads are necessary here.

  const auto num_indices = idx_spans.first.size() + idx_spans.second.size();
  const auto chunk_size = blockDim.x;
  const auto chunk_max = num_indices;
  // one thread per xor_source
  for (unsigned chunk_idx{}; chunk_idx * chunk_size < chunk_max; ++chunk_idx)
{
    __syncthreads();
    // "flat" index, i.e. not yet indexed into appropriate idx_spans array
    const auto flat_idx = chunk_idx * chunk_size + threadIdx.x;
    if (flat_idx < num_indices) {
      auto combo_idx = get_combo_index(flat_idx, idx_spans);
      if (is_source_XOR_compatible(source, combo_idx, xor_src_lists,
              xor_src_list_start_indices, xor_idx_lists,
              xor_idx_list_start_indices, xor_idx_list_sizes, num_idx_lists))
{ is_xor_compat = true;
      }
    }
    __syncthreads();
    if (is_xor_compat) {
      return true;
    }
  }
  return false;
}

__global__ void mark_or_sources_kernel(
    const SourceCompatibilityData* __restrict__ xor_src_lists,
    const index_t* __restrict__ xor_src_list_start_indices,
    const index_t* __restrict__ xor_idx_lists,
    const index_t* __restrict__ xor_idx_list_start_indices,
    const index_t* __restrict__ xor_idx_list_sizes,
    const unsigned num_idx_lists,
    const device::VariationIndices* __restrict__ variation_indices,
    const unsigned num_or_args,
    const device::OrSourceData* __restrict__ or_src_list,
    const unsigned num_or_sources, result_t* __restrict__ results) {
  // for each or_source (one block per source)
  for (unsigned idx{blockIdx.x}; idx < num_or_sources; idx += gridDim.x) {
    __syncthreads();
    const auto& or_source = or_src_list[idx];
    auto result =
        get_smallest_src_index_spans(or_source.src, variation_indices);
    using enum SmallestSpans::ResultCode;
    if (result.code == None) {
      continue;
    }
    if ((result.code == Check)
        && !is_source_XOR_compatible_with_any(or_source.src, result.idx_spans,
            xor_src_lists, xor_src_list_start_indices, xor_idx_lists,
            xor_idx_list_start_indices, xor_idx_list_sizes, num_idx_lists)) {
      continue;
    }
    // result.code is All, or Check and compatibilty check succeeded
    if (!threadIdx.x) {
      results[idx] = 1;
    }
  }
}
*/

/*
auto flat_index(
    const SourceCompatibilityLists& src_list, const SourceIndex src_idx) {
  uint32_t flat{};
  for (size_t i{}; i < src_idx.listIndex; ++i) {
    flat += src_list.at(i).size();
  }
  return flat + src_idx.index;
}

__device__ __host__ auto isSourceXORCompatibleWithAnyXorSource(
    const SourceCompatibilityData& source, const XorSource* xorSources,
    size_t numXorSources) {
  bool compatible = true;
  for (size_t i{}; i < numXorSources; ++i) {
    compatible = source.isXorCompatibleWith(xorSources[i]);
    if (compatible) {
      break;
    }
  }
  return compatible;
}

void check(const SourceCompatibilityLists& src_list, index_t list_index,
    index_t index) {
  constexpr const auto logging = true;
  if constexpr (logging) {
    SourceIndex src_idx{list_index, index};
    char idx_buf[32];
    char buf[64];
    snprintf(buf, sizeof(buf), "%s, flat: %d", src_idx.as_string(idx_buf),
        flat_index(src_list, src_idx));
    auto& source = src_list.at(list_index).at(index);
    source.dump(buf);
    auto compat = isSourceXORCompatibleWithAnyXorSource(
        source, MFD.xorSourceList.data(), MFD.xorSourceList.size());
    std::cerr << "compat: " << compat << std::endl;
  }
}

void dump_xor(int index) {
  const XorSourceList& xorSources = MFD.xorSourceList;
  auto host_index = index;
  const auto& src = xorSources.at(host_index);
  char buf[32];
  snprintf(buf, sizeof(buf), "xor: device(%d) host(%d)", index, host_index);
  src.dump(buf);
}
*/

/*
void run_mark_or_sources_kernel(
    const MergeFilterData& mfd, result_t* device_results) {
  int num_sm;
  hipDeviceGetAttribute(&num_sm, hipDeviceAttributeMultiprocessorCount, 0);
  int threads_per_sm;;
  hipDeviceGetAttribute(&threads_per_sm,
hipDeviceAttributeMaxThreadsPerMultiProcessor, 0); auto block_size = 768; auto
blocks_per_sm = threads_per_sm / block_size; assert(blocks_per_sm * block_size
== threads_per_sm); auto grid_size = num_sm * blocks_per_sm;  // aka blocks
per grid auto shared_bytes = 0;

  dim3 grid_dim(grid_size);
  dim3 block_dim(block_size);
  hipStream_t stream = hipStreamPerThread;
  //hipStreamSynchronize(hipStreamPerThread);
  mark_or_sources_kernel<<<grid_dim, block_dim, shared_bytes, stream>>>(
    mfd.device.src_lists, mfd.device.src_list_start_indices,
    mfd.device.idx_lists, mfd.device.idx_list_start_indices,
    mfd.device.idx_list_sizes, mfd.host.compat_idx_lists.size(),
    mfd.device.variation_indices, mfd.host.or_arg_list.size(),
    mfd.device.or_src_list, mfd.device.num_or_sources, device_results);
}
*/

}  // namespace cm
